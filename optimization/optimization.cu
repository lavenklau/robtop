#include "hip/hip_runtime.h"
#include "optimization.h"
#include "hip/hip_runtime.h"
//#include "device_atomic_functions.hpp"
#include "lib.cuh"
#include "Grid.h"
#include "gpuVector.h"
#include <vector>
#include "templateMatrix.h"
//#include "gpuVector.h"

using namespace culib;
using namespace grid;

extern  __constant__  double gTemplateMatrix[24][24];
extern  __constant__ int* gV2E[8];
extern  __constant__ int* gV2Vfine[27];
extern  __constant__ int* gV2Vcoarse[8];
extern  __constant__ int* gV2V[27];
extern  __constant__ int* gVfine2Vfine[27];
extern  __constant__ int* gV2VfineC[64];// vertex to fine grid element center 
extern  __constant__ int* gVfine2Efine[8];
extern  __constant__ int* gVfine2Effine[8];
extern  __constant__ float power_penalty[1];
extern  __constant__ double* gU[3];
extern  __constant__ double* gF[3];
extern  __constant__ double* gR[3];
extern  __constant__ double* gUworst[3];
extern  __constant__ double* gFworst[3];
extern  __constant__ double* gRfine[3];
extern  __constant__ double* gUcoarse[3];
extern  __constant__ int gGS_num[8];
extern  __constant__ int gmode[1];
extern  __constant__ int* gVflag[1];
extern  __constant__ int* gEflag[1];
extern  __constant__ int gLayerid[1];
extern  __constant__ int gDEBUG[1];

extern __constant__ ScalarT* gT;
extern __constant__ ScalarT* gFT;
extern __constant__ ScalarT* gRTfine;
extern __constant__ ScalarT* gTcoarse;
extern __constant__ ScalarT* gRT;

extern __constant__ double gHeatMatrix[8][8];

extern __device__ void loadTemplateMatrix(volatile double KE[24][24]);

template<int N>
__device__ int gridPos2id(int x, int y, int z) {
	return x + y * N + z * N*N;
}

//  suppose Uworst, Fworst is prepared in U, F
__global__ void computeSensitivity_kernel(int nv, float* rholist, double mu, float* sens) {
	int tid = blockDim.x*blockIdx.x + threadIdx.x;

	__shared__ double KE[24][24];

	loadTemplateMatrix(KE);

	if (tid >= nv) return;

	int vid = tid;

	int vneigh[27];
	for (int i = 0; i < 27; i++) {
		vneigh[i] = gV2V[i][vid];
	}

	// traverse incident elements of vertex
	for (int i = 0; i < 8; i++) {
		double partialSens = 0;

		int eid = gV2E[i][vid];
		if (eid == -1) continue;
		double Ui[3] = { gU[0][vid],gU[1][vid],gU[2][vid] };
		double penal = power_penalty[0] * powf(rholist[eid], power_penalty[0] - 1);

		// compute partial node force (element i's contribution) K_\rho * Uworst on vi
		double KrhoU[3] = { 0. };
		// vertex self id in neihbor element i
		int vi = 7 - i;
		// vertex neighbor id in element i, traverse them and compute the corresponding node force on self
		for (int vj = 0; vj < 8; vj++) {
			int vjlid = gridPos2id<3>(i % 2 + vj % 2, i % 4 / 2 + vj % 4 / 2, i / 4 + vj / 4);
			double Uj[3];
			for (int k = 0; k < 3; k++) Uj[k] = gU[k][vneigh[vjlid]];
			for (int krow = 0; krow < 3; krow++) {
				for (int kcol = 0; kcol < 3; kcol++) {
					KrhoU[krow] += KE[vi * 3 + krow][vj * 3 + kcol] * Uj[kcol];
				}
			}
		}

		for (int k = 0; k < 3; k++) KrhoU[k] *= penal;

#if 0
		// sensitivity  u_worst * dK/drho * u_worst
		for (int k = 0; k < 3; k++) partialSens += Ui[k] * KrhoU[k];

		// sensitivity  - 2 mu * u_worst * dK/drho * K * u_worst
		for (int k = 0; k < 3; k++) partialSens += -2 * mu * KrhoU[k] * gFworst[k][vid];

		// sensitivity  - lambda * N * dK/drho * u_worst
		for (int k = 0; k < 3; k++) {
			partialSens += -KrhoU[k] * gU[k][vid];
		}
#else
		// sensitivity  - u_worst * dK/drho * u_worst
		for (int k = 0; k < 3; k++) partialSens -= Ui[k] * KrhoU[k];

#endif

		atomicAdd(sens + eid, float(partialSens));
	}

}

void computeSensitivity(void) {
	grids[0]->use_grid();
	// now, suppose Uworst, Fworst is prepared, N^T * Lambda is in U,
	// copy Fworst=KUworst to F
	//grids[0]->v3_copy(grids[0]->getWorstForce(), grids[0]->getForce());

	// init sensitivity to zero
	init_array(grids[0]->getSens(), float{ 0 }, grids[0]->n_rho());

	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, grids[0]->n_nodes(), 512);

	computeSensitivity_kernel << <grid_size, block_size >> > (grids[0]->n_nodes(), grids[0]->getRho(), grids[0]->_keyvalues["mu"], grids[0]->getSens());
	hipDeviceSynchronize();
	cuda_error_check;

	// DEBUG
	//grids[0]->sens2matlab("sens");

	// filter sensitivity
	grids[0]->filterSensitivity(params.filter_radius);

	// DEBUG
	grids[0]->sens2matlab("sensfilt");
}


__global__ void trySensMultiplier_kernel(
	int nv, const float* rholist, float* g_sens, float g_thres, float step, float damp, float rhomin, float* newrho) {
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (tid >= nv) return;

	int eid = gV2E[7][tid];

	if (eid == -1) { return; }

	float g = g_sens[eid];

	if (g > 0) g = 0;

	g = abs(g);

	float rhoold = rholist[eid];

	float rhonew = rhoold * powf(g / g_thres, damp);

	rhonew = clamp(rhonew, rhoold - step, rhoold + step);

	rhonew = clamp(rhonew, rhomin, 1.f);

	if (gEflag[0][eid] & grid::Grid::Bitmask::mask_shellelement) rhonew = 1;

	newrho[eid] = rhonew;
}

float updateDensities(float Vgoal) {
	grids[0]->use_grid();

	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, grids[0]->n_nodes(), 512);

	float Vratio = 2;

	float g_thres_low = 0;
	float g_thres_upp = 1;

	// compute old volume ratio
	// double* sum = (double*)grid::Grid::getTempBuf(sizeof(double) * grids[0]->n_rho() / 100);
	double Vold = parallel_sum_d(grids[0]->getRho(), grids[0]->n_rho()) / grids[0]->n_rho();

	// compute maximal sensitivity
	// float* maxdump = (float*)grid::Grid::getTempBuf(sizeof(float)* grids[0]->n_rho() / 100);
	float g_max = parallel_maxabs(grids[0]->getSens(), grids[0]->n_rho());

	g_thres_upp = g_max;

	printf("[sensitivity] max = %f\n", g_max);

	float g_thres = (g_thres_low + g_thres_upp) / 2;

	// iteration counter
	int itn = 0;

	// bisection search sensitivity multiplier
	do  {
		// update sensitivity threshold
		g_thres = (g_thres_low + g_thres_upp) / 2;

		printf("-- searching multiplier g = %4.4e", g_thres);

		float* newrho = (float*)grid::Grid::getTempBuf(sizeof(float)* grids[0]->n_rho());

		// update new rho
		trySensMultiplier_kernel << <grid_size, block_size >> > (
			grids[0]->n_nodes(), grids[0]->getRho(), grids[0]->getSens(), g_thres, params.design_step, params.damp_ratio, params.min_rho, newrho);
		hipDeviceSynchronize();
		cuda_error_check;

		// compute new volume ratio
		Vratio = dump_array_sum(newrho, grids[0]->n_rho()) / grids[0]->n_rho();

		printf(", V = %f  goal %f\n", Vratio, Vgoal);

		if (Vratio > Vgoal) {
			g_thres_low = g_thres;
		}
		else if (Vratio < Vgoal) {
			g_thres_upp = g_thres;
		}
	} while (abs(Vratio - Vgoal) > 1e-4 && itn++ < 30);

	// update densities according to new sensitivity
	trySensMultiplier_kernel << <grid_size, block_size >> > (grids[0]->n_nodes(), grids[0]->getRho(), grids[0]->getSens(), g_thres, params.design_step, params.damp_ratio, params.min_rho, grids[0]->getRho());
	hipDeviceSynchronize();
	cuda_error_check;
	
	return g_thres;
}


extern void matlab_utils_test(void);

void selfTest(void)
{
	printf("-- Self testing...\n");
	using namespace grid;

	std::vector<int> arr(10000, 0xaaaaaaaa);

	BitSAT<int> bits(arr);

	printf("-- host bits total = %d\n", bits.total());

	gBitSAT<int> gbits(bits._bitArray, bits._chunkSat);

	{
		std::vector<int> bitscheck(bits._bitArray.size());
		hipMemcpy(bitscheck.data(), gbits._bitarray, sizeof(int) * bitscheck.size(), hipMemcpyDeviceToHost);
		//printf("%p ->\n", gbits._bitarray);
		//for (int i = 0; i < bitscheck.size(); i++) printf("%d ", bitscheck[i]);
		//printf("\n");
	}

	devArray_t<int*, 1> gcount;
	hipMalloc(&gcount[0], bits._bitArray.size() * BitCount<int>::value * sizeof(int));

	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, bits._bitArray.size() * BitCount<int>::value, 128);

	auto kernel = [=] __device__(int tid) {
		if (blockIdx.x == 0 && threadIdx.x == 0) {
			//printf("_bitarray = %p ; _chunksat = %p\n", gbits._bitarray, gbits._chunksat);
			//printf("firstOne = %d\n", ::firstOne<sizeof(int) * 8>::value);
		}
		gcount[0][tid] = gbits[tid];
	};
	traverse_noret << <grid_size, block_size >> > (bits._bitArray.size() * BitCount<int>::value, kernel);
	hipDeviceSynchronize();
	cuda_error_check;

	std::vector<int> counts(bits._bitArray.size() * BitCount<int>::value);
	hipMemcpy(counts.data(), gcount[0], sizeof(int) * bits._bitArray.size() * BitCount<int>::value, hipMemcpyDeviceToHost);

	bool pass_test = true;
	for (int i = 0; i < counts.size(); i++) {
		//printf("[%d] : %d\n", i, counts[i]);
		if (counts[i] != bits[i]) {
			pass_test = false;
			break;
		}
	}

	gbits.destroy();
	gcount.destroy();

	cuda_error_check;

	matlab_utils_test();

	// test GraftArray
	{
		std::vector<int> hostbuf(10000);
		int* _bufdev;
		hipMalloc(&_bufdev, sizeof(int)*hostbuf.size());
		int baselen = hostbuf.size();
		auto kern = [=] __device__(int tid) {
			int rl = baselen / 125;
			GraftArray<int, 25, 5> p(_bufdev, rl);
			int id[3] = { tid % rl, tid / rl % 5,tid / rl / 5 };
			p[id[2]][id[1]][id[0]] = tid;
		};
		size_t grid_size, block_size;
		make_kernel_param(&grid_size, &block_size, hostbuf.size(), 512);
		traverse_noret << <grid_size, block_size >> > (hostbuf.size(), kern);
		hipDeviceSynchronize();
		cuda_error_check;
		hipMemcpy(hostbuf.data(), _bufdev, sizeof(int) * hostbuf.size(), hipMemcpyDeviceToHost);

		bool fail = false;
		for (int i = 0; i < hostbuf.size(); i++) {
			if (hostbuf[i] != i) {
				fail = true; break;
			}
		}
		if (fail) {
			printf("-- GraftArray test failed\n");
		}
		pass_test &= !fail;

		hipFree(_bufdev);
	}

	if (pass_test) {
		printf("-- Pass test\n");
	}
	else {
		printf("-- Test failed\n");
	}

}

// upload template matrix and power penalty coefficient
void uploadTemplateMatrix(void)
{
	double element_len = grids.elementLength();
	initTemplateMatrix(element_len, gpu_manager, params.youngs_modulu, params.poisson_ratio);
	const double* ke = getTemplateMatrixElements();
	hipMemcpyToSymbol(HIP_SYMBOL(gTemplateMatrix), ke, sizeof(gTemplateMatrix));
	cuda_error_check;

	// upload power penalty
	float power = params.power_penalty;
	hipMemcpyToSymbol(HIP_SYMBOL(power_penalty), &power, sizeof(power_penalty));
	cuda_error_check;

	// upload heat matrix
	const double* pKT = getHeatTemplateMatrixElements();
	hipMemcpyToSymbol(HIP_SYMBOL(gHeatMatrix), pKT, sizeof(gHeatMatrix));
}

void setDEBUG(bool debug)
{
	int a = 0;
	if (debug) a = 1;
	hipMemcpyToSymbol(HIP_SYMBOL(gDEBUG), &a, sizeof(int));
}


__global__ void checkAjointKernel(int n_gsvertices, double mu, devArray_t<double*, 3> vdst) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	if (tid >= n_gsvertices) return;

	for (int i = 0; i < 3; i++) {
		double kku = gF[i][tid];
		double f = vdst[i][tid];
		double fworst = gFworst[i][tid];
		vdst[i][tid] = 2 * fworst - f - 2 * mu * kku;
	}
}

// supporst N^T lambda is in U,  uworst is in Uworst, Kuworst is in Fworst
bool checkAdjointVariable(void) {
	grids[0]->use_grid();
	devArray_t<double*, 3> vdst, uback;
	for (int i = 0; i < 3; i++) {
		hipMalloc(&vdst[i], sizeof(double) * grids[0]->n_gsvertices);
		hipMalloc(&uback[i], sizeof(double) * grids[0]->n_gsvertices);
	}
	// backup adjoint displacement
	grids[0]->v3_copy(grids[0]->getDisplacement(), uback._data);
	// KNlam in vdst
	grids[0]->applyK(grids[0]->getDisplacement(), grids[0]->getForce());
	grids[0]->v3_copy(grids[0]->getForce(), vdst._data);
	grids[0]->resetDirchlet(vdst._data);

	// KKu is in F
	grids[0]->v3_copy(grids[0]->getWorstForce(), grids[0]->getDisplacement());
	grids[0]->applyK(grids[0]->getDisplacement(), grids[0]->getForce());
	grids[0]->resetDirchlet(grids[0]->getForce());
	double mu = grids[0]->_keyvalues["mu"];
	
	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, grids[0]->n_gsvertices, 512);
	checkAjointKernel<<<grid_size,block_size>>>(grids[0]->n_gsvertices, mu, vdst);
	hipDeviceSynchronize();
	cuda_error_check;
	grids[0]->v3_toMatlab("vadj", vdst._data);
	bool vnorm = grids[0]->v3_norm(vdst._data);
	bool passCheck = vnorm < 1e-6;

	grids[0]->v3_copy(uback._data, grids[0]->getDisplacement());

	vdst.destroy();
	uback.destroy();

	return passCheck;
}

__global__ void setSinkNodes_kernel(int nv, int* vflags) {
	size_t vid = blockIdx.x * blockDim.x + threadIdx.x;
	if(vid >= nv) return;
	int flag = vflags[vid];
	if(flag & Grid::mask_supportnodes) {
		flag = flag | Grid::mask_sink_nodes;
	}
	vflags[vid] = flag;
}

// ToDo : set sink node
void setSinkNodes(grid::Grid &grid)
{
	size_t grid_size, block_size;
	int nv = grid.n_gsvertices;
	make_kernel_param(&grid_size, &block_size, grid.n_gsvertices, 512);
	setSinkNodes_kernel<<<grid_size, block_size>>>(nv, grid._gbuf.vBitflag);
	hipDeviceSynchronize();
	cuda_error_check;
}

__global__ void averageNodeT_kernel(int nv, int *vflags, ScalarT *telist) {
	size_t vid = blockIdx.x * blockDim.x + threadIdx.x;
	if(vid >= nv) return;
	int vflag = vflags[vid];
	bool invalid = vflag & Grid::mask_invalid;
	if(!invalid) return;
	int eid = gV2E[7][vid];
	if (eid == -1) return;
	ScalarT sT = 0;
	for (int i = 0; i < 8; i++) {
		int vneioff[3] = {i % 2 + 1, i / 2 % 2 + 1, i / 4 + 1};
		int vn = vneioff[0] + vneioff[1] * 3 + vneioff[2] * 9;
		int vnid = gV2V[vn][vid];
		if(vnid) print_exception;
		auto vt = gT[vnid];
		sT += vt;
	}
	sT /= 8;
	telist[eid] = sT;
}

void averageNodeT(grid::Grid &grid)
{
	grid.use_grid();
	size_t grid_size, block_size;
	int nv = grid.n_gsvertices;
	make_kernel_param(&grid_size, &block_size, grid.n_gsvertices, 512);
	averageNodeT_kernel<<<grid_size, block_size>>>(nv, grid._gbuf.vBitflag, grid._gbuf.te);
	hipDeviceSynchronize();
	cuda_error_check;
}

void testHeatFEM(void) {
	auto& grid = *grids[0];
	grid.use_grid();
	grid.reset_heat_displacement();
	grid.reset_heat_force();
	grid.reset_heat_residual();
	// init Conductivity
	init_array(grid._gbuf.ce, 0.5f, grid.n_gselements);
	// init source
	init_array(grid._gbuf.fT, 1.f, grid.n_gsvertices);
	// choose sink nodes
	setSinkNodes(grid);
	update_heat_stencil();
	solveHeatFEM();
	averageNodeT(grid);
	grids.writeHeat(grids.getPath("te.vdb"));
}