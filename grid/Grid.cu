#include "hip/hip_runtime.h"
#include "Grid.h"
//#include "device_atomic_functions.h"
#include ""
#include "hip/hip_runtime.h"
#include "templateMatrix.h"
#include "lib.cuh"
#include "projection.h"
#include "tictoc.h"
//#define GLM_FORCE_CUDA
//// #define GLM_FORCE_PURE (not needed anymore with recent GLM versions)
//#include <glm/glm.hpp>
//#include "matlab_utils.h"

#define DIRICHLET_DIAGONAL_WEIGHT 1e6f
//#define DIRICHLET_DIAGONAL_WEIGHT 1

using namespace grid;
using namespace culib;

__constant__ double gTemplateMatrix[24][24];
__constant__ int* gV2E[8];
__constant__ int* gV2Vfine[27];
__constant__ int* gV2Vcoarse[8];
__constant__ int* gV2V[27];
__constant__ int* gVfine2Vfine[27];
__constant__ int* gV2VfineC[64];// vertex to fine grid element center 
__constant__ int* gVfine2Efine[8];
__constant__ int* gVfine2Effine[8];
__constant__ float power_penalty[1];
__constant__ double* gU[3];
__constant__ double* gF[3];
__constant__ double* gR[3];
__constant__ double* gUworst[3];
__constant__ double* gFworst[3];
__constant__ double* gRfine[3];
__constant__ double* gUcoarse[3];
__constant__ int gGS_num[8];
__constant__ int gmode[1];
__constant__ int* gVflag[1];
__constant__ int* gEflag[1];
__constant__ int gLayerid[1];
__constant__ int gDEBUG[1];

__constant__ ScalarT* gT;
__constant__ ScalarT* gFT;
__constant__ ScalarT* gRT;

extern __constant__ double* gLoadtangent[2][3];
extern __constant__ double* gLoadnormal[3];

extern gBitSAT<unsigned int> vid2loadid;

void Grid::use_grid(void)
{
	hipMemcpyToSymbol(HIP_SYMBOL(gV2V), _gbuf.v2v, sizeof(gV2V));
	hipMemcpyToSymbol(HIP_SYMBOL(gV2Vfine), _gbuf.v2vfine, sizeof(gV2Vfine));
	hipMemcpyToSymbol(HIP_SYMBOL(gV2Vcoarse), _gbuf.v2vcoarse, sizeof(gV2Vcoarse));
	hipMemcpyToSymbol(HIP_SYMBOL(gV2E), _gbuf.v2e, sizeof(gV2E));
	hipMemcpyToSymbol(HIP_SYMBOL(gV2VfineC), _gbuf.v2vfinecenter, sizeof(gV2VfineC));
	hipMemcpyToSymbol(HIP_SYMBOL(gU), _gbuf.U, sizeof(gU));
	hipMemcpyToSymbol(HIP_SYMBOL(gF), _gbuf.F, sizeof(gF));
	hipMemcpyToSymbol(HIP_SYMBOL(gR), _gbuf.R, sizeof(gR));
	hipMemcpyToSymbol(HIP_SYMBOL(gUworst), _gbuf.Uworst, sizeof(gUworst));
	hipMemcpyToSymbol(HIP_SYMBOL(gFworst), _gbuf.Fworst, sizeof(gFworst));
	hipMemcpyToSymbol(HIP_SYMBOL(gGS_num), gs_num, sizeof(gGS_num));
	hipMemcpyToSymbol(HIP_SYMBOL(gVflag), &_gbuf.vBitflag, sizeof(gVflag));
	hipMemcpyToSymbol(HIP_SYMBOL(gEflag), &_gbuf.eBitflag, sizeof(gEflag));
	hipMemcpyToSymbol(HIP_SYMBOL(gLayerid), &_layer, sizeof(gLayerid));

	if (fineGrid != nullptr) {
		hipMemcpyToSymbol(HIP_SYMBOL(gVfine2Vfine), fineGrid->_gbuf.v2v, sizeof(gVfine2Vfine));
		hipMemcpyToSymbol(HIP_SYMBOL(gVfine2Efine), fineGrid->_gbuf.v2e, sizeof(gVfine2Efine));
		hipMemcpyToSymbol(HIP_SYMBOL(gRfine), fineGrid->_gbuf.R, sizeof(gRfine));
	}
	if (coarseGrid != nullptr) {
		hipMemcpyToSymbol(HIP_SYMBOL(gUcoarse), coarseGrid->_gbuf.U, sizeof(gUcoarse));
	}
	//hipDeviceSynchronize();
	cuda_error_check;
}

__device__ bool isValidNode(int vid) {
	return gV2V[13][vid] != -1;
}

__device__ void loadTemplateMatrix(volatile double KE[24][24]) {
	int i = threadIdx.x / 24;
	int j = threadIdx.x % 24;
	if (i < 24) {
		KE[i][j] = gTemplateMatrix[i][j];
	}
	int nfill = blockDim.x;
	while (nfill < 24 * 24) {
		int kid = nfill + threadIdx.x;
		i = kid / 24;
		j = kid % 24;
		if (i < 24) {
			KE[i][j] = gTemplateMatrix[i][j];
		}
		nfill += blockDim.x;
	}
	__syncthreads();
}

__device__ void loadNeighborNodesAndFlags(int vid, int v2v[27], bool vfix[27], bool vload[27]) {
	int* pflag = gVflag[0];
	for (int i = 0; i < 27; i++) {
		v2v[i] = gV2V[i][vid];
		if (v2v[i] != -1) {
			int flag = pflag[v2v[i]];
			vfix[i] = flag & grid::Grid::Bitmask::mask_supportnodes;
			vload[i] = flag & grid::Grid::Bitmask::mask_loadnodes;
		}
	}
}

__device__ void loadNeighborNodes(int vid, int v2v[27]) {
	for (int i = 0; i < 27; i++) { v2v[i] = gV2V[i][vid]; }
}

/*
	//rxcoarse[32(27)][9][nv]
	rxcoarse[27][9][nv]
*/
template<int BlockSize = 32 * 9>
__global__ void restrict_stencil_dyadic_kernel(int nv_coarse, double* rxcoarse_, int nv_fine, double* rxfine_) {
	size_t tid = blockDim.x*blockIdx.x + threadIdx.x;
	int ke_id = tid / nv_coarse;
	int vid = tid % nv_coarse;

	if (ke_id >= 9) return;

	GraftArray<double, 27, 9> rxCoarse(rxcoarse_, nv_coarse);
	GraftArray<double, 27, 9> rxFine(rxfine_, nv_fine);

	//__shared__ double coarseStencil[27][BlockSize / 32][32];
	//initSharedMem(&coarseStencil[0][0][0], sizeof(coarseStencil) / sizeof(double));
	double coarseStencil[27] = { 0. };

	int warpid = threadIdx.x / 32;
	int warptid = threadIdx.x % 32;

	double w[4] = { 1.0,1.0 / 2,1.0 / 4,1.0 / 8 };
	for (int i = 0; i < 27; i++) {
		int neipos[3] = { i % 3 + 1 ,i % 9 / 3 + 1 ,i / 9 + 1 };

		int wneighpos[3] = { abs(neipos[0] - 2),abs(neipos[1] - 2),abs(neipos[2] - 2) };

		if (wneighpos[0] >= 2 || wneighpos[1] >= 2 || wneighpos[2] >= 2) continue;

		double weight = w[wneighpos[0] + wneighpos[1] + wneighpos[2]];

		int vn = gV2Vfine[i][vid];

		if (vn == -1) continue;

		// traverse fine stencil component (each neighbor vertex has a component)
		for (int j = 0; j < 27; j++) {

			double kij = rxFine[j][ke_id][vn] * weight;

			// DEBUG
			if (gVfine2Vfine[j][vn] == -1) { if (kij != 0) { printf("-- error on stencil 1\n"); } continue; }

			int vjpos[3] = { neipos[0] + j % 3 - 1 ,neipos[1] + j % 9 / 3 - 1 ,neipos[2] + j / 9 - 1 };

			// traverse coarse vertices to scatter the stencil component to them
			for (int vsplit = 0; vsplit < 27; vsplit++) {
				int vsplitpos[3] = { vsplit % 3 * 2, vsplit % 9 / 3 * 2, vsplit / 9 * 2 };
				int wsplitpos[3] = { abs(vsplitpos[0] - vjpos[0]), abs(vsplitpos[1] - vjpos[1]), abs(vsplitpos[2] - vjpos[2]) };
				if (wsplitpos[0] >= 2 || wsplitpos[1] >= 2 || wsplitpos[2] >= 2) continue;
				double wsplit = w[wsplitpos[0] + wsplitpos[1] + wsplitpos[2]];
				coarseStencil[vsplit] += wsplit * kij;
			}
		}
	}

	for (int i = 0; i < 27; i++) {
		//rxCoarse[i][ke_id][vid] = coarseStencil[i][warpid][warpid];
		rxCoarse[i][ke_id][vid] = coarseStencil[i];
	}
}

// on the fly assembly
template<int BlockSize = 32 * 9>
__global__ void restrict_stencil_dyadic_OTFA_kernel(int nv_coarse, double* rxcoarse_, int nv_fine, float* rhofine) {
	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	//__shared__ int restrict_elements[64];
	__shared__ double KE[24][24];

	// load template matrix from constant memory to shared memory
	loadTemplateMatrix(KE);

	int warpid = threadIdx.x / 32;
	int warptid = threadIdx.x % 32;

	int ke_id = tid / nv_coarse;
	int vid = tid % nv_coarse;

	if (ke_id >= 9) return;

	GraftArray<double, 27, 9> rxCoarse(rxcoarse_, nv_coarse);

	//__shared__ double coarseStencil[27][BlockSize / 32][32];
	//initSharedMem(&coarseStencil[0][0][0], sizeof(coarseStencil) / sizeof(double));

	double coarseStencil[27] = { 0. };

	
	//for (int i = 0; i < 27; i++) {
	//	coarseStencil[i][warpid][warptid] = 0;
	//}
	

	// reorder K3 in row major order 
	int k3row = ke_id / 3;
	int k3col = ke_id % 3;

	double w[4] = { 1.0,1.0 / 2,1.0 / 4,1.0 / 8 };
	double kc[27] = { 0. };

	int ebit[2] = { 0 };

	float power = power_penalty[0];

	// traverse neighbor nodes on fine grid
	for (int i = 0; i < 27; i++) {
		int neipos[3] = { i % 3 + 1 ,i % 9 / 3 + 1 ,i / 9 + 1 };

		int vn = gV2Vfine[i][vid];

		if (vn == -1) continue;

		// traverse the neighbor element of each neighbor nodes
		for (int j = 0; j < 8; j++) {
			int epos[3] = { neipos[0] + j % 2 - 1,neipos[1] + j % 4 / 2 - 1,neipos[2] + j / 4 - 1 };
			int eposid = epos[0] + epos[1] * 4 + epos[2] * 16;
			if (read_gbit(ebit, eposid)) continue;
			set_gbit(ebit, eposid);
			float rho_p = 0;
			int eid = gVfine2Efine[j][vn];
			if (eid == -1) continue;
			rho_p = powf(rhofine[eid], power);
			// traverse vertex of neighbor elements (rows of element matrix)
			for (int vi = 0; vi < 8; vi++) {
				int vipos[3] = { epos[0] + vi % 2,epos[1] + vi % 4 / 2,epos[2] + vi / 4 };
				int wipos[3] = { abs(vipos[0] - 2) , abs(vipos[1] - 2) , abs(vipos[2] - 2) };
				if (wipos[0] >= 2 || wipos[1] >= 2 || wipos[2] >= 2) continue;
				int wiid = wipos[0] + wipos[1] + wipos[2];
				if (wiid >= 4) continue;
				double wi_p = w[wiid] * rho_p;

				// traverse another vertex of neighbor element (cols of element matrix), compute Ke 3x3
				for (int vj = 0; vj < 8; vj++) {
					int vjpos[3] = { epos[0] + vj % 2,epos[1] + vj % 4 / 2,epos[2] + vj / 4 };
					double ke = 0;
					double wk = wi_p * KE[vi * 3 + k3row][vj * 3 + k3col];

					// scatter 3x3 Ke to coarse nodes, traverse coarse nodes
					for (int vsplit = 0; vsplit < 27; vsplit++) {
						int vsplitpos[3] = { vsplit % 3 * 2, vsplit % 9 / 3 * 2, vsplit / 9 * 2 };
						int wspos[3] = { abs(vsplitpos[0] - vjpos[0]), abs(vsplitpos[1] - vjpos[1]), abs(vsplitpos[2] - vjpos[2]) };
						if (wspos[0] >= 2 || wspos[1] >= 2 || wspos[2] >= 2) continue;
						int wsid = wspos[0] + wspos[1] + wspos[2];
						double wkw = wk * w[wsid];
						coarseStencil[vsplit] += wkw;
					}
				}
			}
		}

	}

	for (int i = 0; i < 27; i++) {
		//rxCoarse[i][ke_id][vid] = coarseStencil[i][warpid][warptid];
		rxCoarse[i][ke_id][vid] = coarseStencil[i];
	}
}

void HierarchyGrid::restrict_stencil_dyadic(Grid& dstcoarse, Grid& srcfine)
{
	dstcoarse.use_grid();
	size_t grid_size, block_size;
	constexpr int BlockSize = 32 * 6;
	if (dstcoarse._layer == 0 && srcfine._layer == 1) {
		make_kernel_param(&grid_size, &block_size, dstcoarse.n_gsvertices * 9, BlockSize);
		restrict_stencil_dyadic_OTFA_kernel<BlockSize> << <grid_size, block_size >> > (dstcoarse.n_gsvertices, dstcoarse._gbuf.rxStencil, srcfine.n_gsvertices, dstcoarse._gbuf.rho_e);
		hipDeviceSynchronize();
		cuda_error_check;
	}
	else {
		make_kernel_param(&grid_size, &block_size, dstcoarse.n_gsvertices * 9, BlockSize);
		restrict_stencil_dyadic_kernel<BlockSize> << <grid_size, block_size >> > (dstcoarse.n_gsvertices, dstcoarse._gbuf.rxStencil, srcfine.n_gsvertices, srcfine._gbuf.rxStencil);
		hipDeviceSynchronize();
		cuda_error_check;
	}
}

// on the fly assembly
template<int BlockSize = 32 * 9>
__global__ void restrict_stencil_nondyadic_OTFA_NS_kernel(int nv_coarse, double* rxcoarse_, int nv_fine, float* rhofine, int* vfineflag) {
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	int warpid = threadIdx.x / 32;
	int warptid = threadIdx.x % 32;


	GraftArray<double, 27, 9> rxCoarse(rxcoarse_, nv_coarse);

	__shared__ double KE[24][24];
	__shared__ double W[4][4][4];
	//__shared__ double coarseStencil[27][BlockSize / 32][32];

	// load template matrix from constant memory to shared memory
	loadTemplateMatrix(KE);

	// compute weight
	if (threadIdx.x < 64) {
		int i = threadIdx.x % 4;
		int j = threadIdx.x % 16 / 4;
		int k = threadIdx.x / 16;
		W[k][j][i] = (4 - i)*(4 - j)*(4 - k) / 64.f;
	}
	__syncthreads();
	
	// init coarseStencil
	//initSharedMem(&coarseStencil[0][0][0], sizeof(coarseStencil) / sizeof(double));
	double coarseStencil[27] = { 0. };

	int ke_id = tid / nv_coarse;

	int vid = tid % nv_coarse;

	if (ke_id >= 9) return;

	//int flagword = vcoarseflag[vid];

	//if (flagword & Grid::Bitmask::mask_invalid) return;

	// reorder K3 in row major order
	int k3row = ke_id / 3;
	int k3col = ke_id % 3;

	float power = power_penalty[0];

	// traverse neighbor nodes of fine element center (which is the vertex on fine fine grid)
	for (int i = 0; i < 64; i++) {
		int i2[3] = { (i % 4) * 2 + 1 ,(i % 16 / 4) * 2 + 1 ,(i / 16) * 2 + 1 };
		//int m2 = i2[0] + i2[1] + i2[2] - 3;

		// get fine element center vertex
		int vn = gV2VfineC[i][vid];

		if (vn == -1) continue;

		// should traverse 7x7x7 neigbor nodes, and sum their weighted stencil, to reduce bandwidth, we traverse 8x8x8 elements 
		// traverse the neighbor fine fine element of this vertex and assembly the element matrices
		for (int j = 0; j < 8; j++) {
			int efineid = gVfine2Efine[j][vn];

			if (efineid == -1) continue;

			float rho_p = powf(rhofine[efineid], power);

			int epos[3] = { i2[0] + j % 2 - 1,i2[1] + j % 4 / 2 - 1,i2[2] + j / 4 - 1 };

			// traverse the vertex of neighbor element (rows of element matrix), compute the weight on this vertex 
			for (int ki = 0; ki < 8; ki++) {
				int vipos[3] = { epos[0] + ki % 2,epos[1] + ki % 4 / 2,epos[2] + ki / 4 };
				int wipos[3] = { abs(vipos[0] - 4),abs(vipos[1] - 4),abs(vipos[2] - 4) };
				if (wipos[0] >= 4 || wipos[1] >= 4 || wipos[2] >= 4) continue;
				double w_ki = W[wipos[0]][wipos[1]][wipos[2]] * rho_p;

				// traverse another vertex of neighbor element (cols of element matrix), get the 3x3 Ke and multiply the row weights
				for (int kj = 0; kj < 8; kj++) {
					int kjpos[3] = { epos[0] + kj % 2 , epos[1] + kj % 4 / 2 , epos[2] + kj / 4 };
					double wk = w_ki * KE[ki * 3 + k3row][kj * 3 + k3col];
					//  the weighted element matrix should split to coarse vertex, traverse the coarse vertices and split 3x3 Ke to coarse vertex by splitting weights
					for (int vsplit = 0; vsplit < 27; vsplit++) {
						int vsplitpos[3] = { vsplit % 3 * 4, vsplit % 9 / 3 * 4,vsplit / 9 * 4 };
						int wjpos[3] = { abs(vsplitpos[0] - kjpos[0]), abs(vsplitpos[1] - kjpos[1]), abs(vsplitpos[2] - kjpos[2]) };
						if (wjpos[0] >= 4 || wjpos[1] >= 4 || wjpos[2] >= 4) continue;
						double wkw = wk * W[wjpos[0]][wjpos[1]][wjpos[2]];
						coarseStencil[vsplit]/*[warpid][warptid]*/ += wkw;
					}
				}
			}
		}
	}

	for (int i = 0; i < 27; i++) {
		rxCoarse[i][ke_id][vid] = coarseStencil[i]/*[warpid][warptid]*/;
	}
}

// on the fly assembly
template<int BlockSize = 32 * 9>
__global__ void restrict_stencil_nondyadic_OTFA_WS_kernel(int nv_coarse, double* rxcoarse_, int nv_fine, float* rhofine, int* vfineflag) {
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	int warpid = threadIdx.x / 32;
	int warptid = threadIdx.x % 32;


	GraftArray<double, 27, 9> rxCoarse(rxcoarse_, nv_coarse);

	__shared__ double KE[24][24];
	__shared__ double W[4][4][4];
	//__shared__ double coarseStencil[27][BlockSize / 32][32];

	// load template matrix from constant memory to shared memory
	loadTemplateMatrix(KE);

	// compute weight
	if (threadIdx.x < 64) {
		int i = threadIdx.x % 4;
		int j = threadIdx.x % 16 / 4;
		int k = threadIdx.x / 16;
		W[k][j][i] = (4 - i)*(4 - j)*(4 - k) / 64.f;
	}
	__syncthreads();
	
	// init coarseStencil
	//initSharedMem(&coarseStencil[0][0][0], sizeof(coarseStencil) / sizeof(double));
	double coarseStencil[27] = { 0. };

	int ke_id = tid / nv_coarse;

	int vid = tid % nv_coarse;

	if (ke_id >= 9) return;

	//int flagword = vcoarseflag[vid];

	//if (flagword & Grid::Bitmask::mask_invalid) return;

	// reorder K3 in row major order
	int k3row = ke_id / 3;
	int k3col = ke_id % 3;

	float power = power_penalty[0];

	// traverse neighbor nodes of fine element center (which is the vertex on fine fine grid)
	for (int i = 0; i < 64; i++) {
		int i2[3] = { (i % 4) * 2 + 1 ,(i % 16 / 4) * 2 + 1 ,(i / 16) * 2 + 1 };
		//int m2 = i2[0] + i2[1] + i2[2] - 3;

		// get fine element center vertex
		int vn = gV2VfineC[i][vid];

		if (vn == -1) continue;

		// should traverse 7x7x7 neigbor nodes, and sum their weighted stencil, to reduce bandwidth, we traverse 8x8x8 elements 
		// traverse the neighbor fine fine element of this vertex and assembly the element matrices
		for (int j = 0; j < 8; j++) {
			int efineid = gVfine2Efine[j][vn];

			if (efineid == -1) continue;

			float rho_p = powf(rhofine[efineid], power);

			int epos[3] = { i2[0] + j % 2 - 1,i2[1] + j % 4 / 2 - 1,i2[2] + j / 4 - 1 };

			// prefecth the flag of eight vertex
			bool vfix[8];
			for (int k = 0; k < 8; k++) {
				int vklid = j % 2 + k % 2 +
					(j / 2 % 2 + k / 2 % 2) * 3 +
					(j / 4 + k / 4) * 9;
				int vkvid = gVfine2Vfine[vklid][vn];
				if (vkvid == -1)printf("-- error in stencil restriction\n");
				int vkflag = vfineflag[vkvid];
				vfix[k] = vkflag & Grid::Bitmask::mask_supportnodes;
			}

			// traverse the vertex of neighbor element (rows of element matrix), compute the weight on this vertex 
			for (int ki = 0; ki < 8; ki++) {
				int vipos[3] = { epos[0] + ki % 2,epos[1] + ki % 4 / 2,epos[2] + ki / 4 };
				int wipos[3] = { abs(vipos[0] - 4),abs(vipos[1] - 4),abs(vipos[2] - 4) };
				if (wipos[0] >= 4 || wipos[1] >= 4 || wipos[2] >= 4) continue;
				double wi = W[wipos[0]][wipos[1]][wipos[2]];
				double w_ki = wi * rho_p;

				// traverse another vertex of neighbor element (cols of element matrix), get the 3x3 Ke and multiply the row weights
				for (int kj = 0; kj < 8; kj++) {
					int kjpos[3] = { epos[0] + kj % 2 , epos[1] + kj % 4 / 2 , epos[2] + kj / 4 };
					double wk = w_ki * KE[ki * 3 + k3row][kj * 3 + k3col];
									
					if (vfix[kj] || vfix[ki]) {
						wk = 0;
						if (ki == kj && k3row == k3col) {
							wk = wi * DIRICHLET_DIAGONAL_WEIGHT;
						}
					}

					//  the weighted element matrix should split to coarse vertex, traverse the coarse vertices and split 3x3 Ke to coarse vertex by splitting weights
					for (int vsplit = 0; vsplit < 27; vsplit++) {
						int vsplitpos[3] = { vsplit % 3 * 4, vsplit % 9 / 3 * 4,vsplit / 9 * 4 };
						int wjpos[3] = { abs(vsplitpos[0] - kjpos[0]), abs(vsplitpos[1] - kjpos[1]), abs(vsplitpos[2] - kjpos[2]) };
						if (wjpos[0] >= 4 || wjpos[1] >= 4 || wjpos[2] >= 4) continue;
						double wkw = wk * W[wjpos[0]][wjpos[1]][wjpos[2]];
						coarseStencil[vsplit]/*[warpid][warptid]*/ += wkw;
					}
				}
			}
		}
	}

	for (int i = 0; i < 27; i++) {
		rxCoarse[i][ke_id][vid] = coarseStencil[i]/*[warpid][warptid]*/;
	}
}

void HierarchyGrid::restrict_stencil_nondyadic(Grid& dstcoarse, Grid& srcfine)
{
	if (dstcoarse._layer != 2 || srcfine._layer != 0) {
		std::cout << "\033[31m" << "Non dyadic restriction is only applied on finest grid" << "\033[0m" << std::endl;
	}

	dstcoarse.use_grid();

	constexpr int BlockSize = 32 * 4;
	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, dstcoarse.n_gsvertices * 9, BlockSize);
	if (_mode == no_support_constrain_force_direction || _mode == no_support_free_force) {
		restrict_stencil_nondyadic_OTFA_NS_kernel<BlockSize> << <grid_size, block_size >> > (dstcoarse.n_gsvertices, dstcoarse._gbuf.rxStencil, srcfine.n_gsvertices, srcfine._gbuf.rho_e, srcfine._gbuf.vBitflag);
	}
	else if (_mode == with_support_constrain_force_direction || _mode == with_support_free_force) {
		restrict_stencil_nondyadic_OTFA_WS_kernel<BlockSize> << <grid_size, block_size >> > (dstcoarse.n_gsvertices, dstcoarse._gbuf.rxStencil, srcfine.n_gsvertices, srcfine._gbuf.rho_e, srcfine._gbuf.vBitflag);
	}
	hipDeviceSynchronize();
	cuda_error_check;
}

void HierarchyGrid::restrict_stencil(Grid& dstcoarse, Grid& srcfine)
{
	if (dstcoarse.is_dummy()) return;
	if (dstcoarse._layer == 0) return;

	init_array(dstcoarse._gbuf.rxStencil, double{ 0 }, 27 * 9 * dstcoarse.n_gsvertices);

	if (_setting.skiplayer1 && dstcoarse._layer == 2 && srcfine._layer == 0) {
		restrict_stencil_nondyadic(dstcoarse, srcfine);
	}
	else {
		if (dstcoarse._layer - srcfine._layer != 1) {
			printf("\033[31mOnly Support stencil restriction between neighbor layers!\033[0m\n");
			throw std::runtime_error("");
		}
		restrict_stencil_dyadic(dstcoarse, srcfine);
	}
}

void Grid::compute_gscolor(gpu_manager_t& gm, BitSAT<unsigned int>& vbit, BitSAT<unsigned int>& ebit, int vreso, int* vbitflaghost, int* ebitflaghost)
{
	int nv = vbit.total();
	int ne = ebit.total();
	int* vbitflagdevice = nullptr;
	int* ebitflagdevice = nullptr;
	int nvword = vbit._bitArray.size();
	int neword = ebit._bitArray.size();

	// build device SAT 
	gBitSAT<unsigned int> gvsat(vbit._bitArray, vbit._chunkSat);
	gBitSAT<unsigned int> gesat(ebit._bitArray, ebit._chunkSat);

	// copy bit flag to device 
	hipMalloc(&vbitflagdevice, nv * sizeof(int));
	hipMalloc(&ebitflagdevice, ne * sizeof(int));
	hipMemcpy(vbitflagdevice, vbitflaghost, sizeof(int) * nv, hipMemcpyHostToDevice);
	hipMemcpy(ebitflagdevice, ebitflaghost, sizeof(int) * ne, hipMemcpyHostToDevice);

	auto vkernel = [=] __device__(int tid) {
		// set vertex gs color  
		unsigned int word = gvsat._bitarray[tid];
		int vid = gvsat._chunksat[tid];
		int vreso2 = vreso * vreso;
		int nvbit = vreso2 * vreso;
		if (word != 0) {
			for (int ji = 0; ji < sizeof(unsigned int) * 8; ji++) {
				if (!read_gbit(word, ji)) continue;
				int vbitid = tid * BitCount<unsigned int>::value + ji;
				if (vbitid >= nvbit) break;
				int pos[3] = { vbitid % vreso, (vbitid % vreso2) / vreso, vbitid / vreso2 };
				int m2 = pos[0] % 2 + pos[1] % 2 * 2 + pos[2] % 2 * 4;
				// set vertex gs color id
				int bitword = vbitflagdevice[vid];
				bitword &= ~(int)Bitmask::mask_gscolor;
				bitword |= (m2 << Bitmask::offset_gscolor);
				vbitflagdevice[vid] = bitword;
				vid++;
			}
		}
		
		// set element gs color
		if (tid >= neword) return;
		word = gesat._bitarray[tid];
		if (word == 0) return;
		int eid = gesat._chunksat[tid];
		int ereso = vreso - 1;
		int ereso2 = ereso * ereso;
		int nebit = ereso * ereso2;
		for (int ji = 0; ji < BitCount<unsigned int>::value; ji++) {
			if (!read_gbit(word, ji)) continue;
			int ebitid = tid * BitCount<unsigned int>::value + ji;
			if (ebitid >= nebit) break;
			int pos[3] = { ebitid % ereso, (ebitid % ereso2) / ereso, ebitid / ereso2 };
			int m2 = pos[0] % 2 + pos[1] % 2 * 2 + pos[2] % 2 * 4;
			int bitword = ebitflagdevice[eid];
			bitword &= ~(int)Bitmask::mask_gscolor;
			bitword |= (m2 << Bitmask::offset_gscolor);
			ebitflagdevice[eid] = bitword;
			eid++;
		}
	};
	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, vbit._bitArray.size(), 512);
	traverse_noret << <grid_size, block_size >> > (vbit._bitArray.size(), vkernel);
	hipDeviceSynchronize();
	cuda_error_check;

	hipMemcpy(vbitflaghost, vbitflagdevice, sizeof(int) * nv, hipMemcpyDeviceToHost);
	hipMemcpy(ebitflaghost, ebitflagdevice, sizeof(int) * ne, hipMemcpyDeviceToHost);

	gvsat.destroy();
	gesat.destroy();
	hipFree(vbitflagdevice);
	hipFree(ebitflagdevice);
}

void* Grid::getTempBuf(size_t requre)
{
	size_t req_size = snippet::Round<512>(requre);
	if (_tmp_buf == nullptr) {
		hipMalloc(&_tmp_buf, req_size);
		_tmp_buf_size = req_size;
	}
	if (_tmp_buf_size < req_size) {
		hipFree(_tmp_buf);
		_tmp_buf_size = snippet::Round<512>(req_size);
		hipMalloc(&_tmp_buf, req_size);
	}
	return _tmp_buf;
}


void Grid::clearBuf(void)
{
	hipFree(_tmp_buf);
}

void Grid::lexico2gsorder_g(int* idmap, int n_id, int* ids, int n_mapid, int* mapped_ids, int* valuemap /*= nullptr*/)
{
	int* pid = ids;
	int* old_ptr;
	if (ids == mapped_ids) {
		old_ptr = (int*)getTempBuf(sizeof(int)*n_id);
		hipMemcpy(old_ptr, ids, sizeof(int) * n_id, hipMemcpyDeviceToDevice);
		pid = old_ptr;
	}
	init_array(mapped_ids, -1, n_mapid);
	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, n_id, 512);
	auto permut = [=] __device__(int tid) {
		int newvalue = pid[tid];
		if (valuemap != nullptr) {
			if (newvalue != -1) {
				newvalue = valuemap[newvalue];
			}
		}
		if (idmap != nullptr) {
			mapped_ids[idmap[tid]] = newvalue;
		}
		else {
			mapped_ids[tid] = newvalue;
		}
	};
	traverse_noret << <grid_size, block_size >> > (n_id, permut);
	hipDeviceSynchronize();
	cuda_error_check;
	
}

template<int BlockSize = 32 * 13>
__global__ void gs_relax_kernel(int n_vgstotal, int nv_gsset, double* rxstencil, int gs_offset) {
	GraftArray<double, 27, 9> stencil(rxstencil, n_vgstotal);
	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	//int mode = gmode[0];

	__shared__ double sumAu[3][13][32];
	int warpId = threadIdx.x / 32;
	int node_id_in_block = threadIdx.x % 32;
	int workId = node_id_in_block;
	int gs_vertex_id = blockIdx.x * 32 + node_id_in_block;

	int offset = gs_offset;

	double Au[3] = { 0.f,0.f,0.f };

	int node_id;

	int flag;

	bool invalid_node = true;

	if (gs_vertex_id < nv_gsset) {
		node_id = offset + gs_vertex_id;
		flag = gVflag[0][node_id];
		invalid_node = flag & Grid::Bitmask::mask_invalid;
		if (invalid_node) goto _blockSum;
		for (auto i : { 0,14 }) {
			double displacement[3];
			int neigh_th = warpId + i;
			int neigh = gV2V[neigh_th][node_id];	
			if (neigh == -1) continue;

			for (int j = 0; j < 3; j++) displacement[j] = gU[j][neigh];

			// K3 is ordered in row major 
			// traverse rows 
			for (int j = 0; j < 3; j++) {
				int jrows = j * 3;
				// traverse columns, dot u 
				for (int k = 0; k < 3; k++) {
					Au[j] += stencil[neigh_th][jrows + k][node_id] * displacement[k];
				}
			}

		}
	}

_blockSum:

	for (int i = 0; i < 3; i++) {
		sumAu[i][warpId][node_id_in_block] = Au[i];
	}
	__syncthreads();

	// gather all part
	if (warpId < 7) {
		for (int i = 0; i < 3; i++) {
			int addId = warpId + 7;
			if (addId < 13) {
				sumAu[i][warpId][node_id_in_block] += sumAu[i][addId][node_id_in_block];
			}
		}
	}
	__syncthreads();
	if (warpId < 4) {
		for (int i = 0; i < 3; i++) {
			int addId = warpId + 4;
			if (addId < 7) {
				sumAu[i][warpId][node_id_in_block] += sumAu[i][addId][node_id_in_block];
			}
		}
	}
	__syncthreads();
	if (warpId < 2) {
		for (int i = 0; i < 3; i++) {
			int addId = warpId + 2;
			sumAu[i][warpId][node_id_in_block] += sumAu[i][addId][node_id_in_block];
		}
	}
	__syncthreads();
	if (warpId < 1) {
		for (int i = 0; i < 3; i++) {
			int addId = warpId + 1;
			Au[i] = sumAu[i][warpId][node_id_in_block] + sumAu[i][addId][node_id_in_block];
		}
	}
	//__syncthreads();

	if (gs_vertex_id < nv_gsset && !invalid_node) {
		double node_sum = 0;

		double displacement[3] = { 0. }; int	rowOffset = 0;

		if (warpId == 0) {
			for (int i = 0; i < 3; i++) displacement[i] = gU[i][node_id];
			node_sum = stencil[13][rowOffset + 1][node_id] * displacement[1] + stencil[13][rowOffset + 2][node_id] * displacement[2];
			displacement[0] = (gF[0][node_id] - Au[0] - node_sum) / stencil[13][0][node_id];
			gU[0][node_id] = displacement[0];

			rowOffset += 3;
			node_sum = stencil[13][rowOffset + 0][node_id] * displacement[0] + stencil[13][rowOffset + 2][node_id] * displacement[2];
			displacement[1] = (gF[1][node_id] - Au[1] - node_sum) / stencil[13][rowOffset + 1][node_id];
			gU[1][node_id] = displacement[1];

			rowOffset += 3;
			node_sum = stencil[13][rowOffset + 0][node_id] * displacement[0] + stencil[13][rowOffset + 1][node_id] * displacement[1];
			displacement[2] = (gF[2][node_id] - Au[2] - node_sum) / stencil[13][rowOffset + 2][node_id];
			gU[2][node_id] = displacement[2];
		}
	}
}

// map 32 vertices to 8 warp, each warp use specific neighbor element (density rho_i)
template<int BlockSize = 32 * 8>
__global__ void gs_relax_OTFA_NS_kernel(int nv_gs, int gs_offset, float* rholist) {
	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	//int mode = gmode[0];

	__shared__ double KE[24][24];

	__shared__ double sumKeU[3][4][32];

	__shared__ double sumS[9][4][32];

	// load template matrix from constant memory to shared memory
	loadTemplateMatrix(KE);

	int warpId = threadIdx.x / 32;
	int warpTid = threadIdx.x % 32;

	double KeU[3] = { 0. };
	double S[9] = { 0. };
	double* pU[3] = { gU[0],gU[1],gU[2] };

	bool invalid_node = false;
	// the id in a gs subset
	int vid = blockIdx.x * 32 + warpTid;

	// the id in total node set
	vid += gs_offset;

	int vi = 7 - warpId;
	double penalty = 0;
	int eid;

	int flag = gVflag[0][vid];
	invalid_node |= flag & Grid::Bitmask::mask_invalid;

	if (invalid_node) goto _blocksum;

	eid = gV2E[warpId][vid];

	if (eid != -1)
		penalty = powf(rholist[eid], power_penalty[0]);
	else
		goto _blocksum;

	if (gV2V[13][vid] == -1) {
		invalid_node = true;
		goto _blocksum;
	}

	// compute KU and S 
	for (int vj = 0; vj < 8; vj++) {
		// vjpos = epos + vjoffset
		int vjpos[3] = {
			vj % 2 + warpId % 2,
			vj % 4 / 2 + warpId % 4 / 2,
			vj / 4 + warpId / 4
		};
		int vj_lid = vjpos[0] + vjpos[1] * 3 + vjpos[2] * 9;
		int vj_vid = gV2V[vj_lid][vid];
		if (vj_vid == -1) continue;
		double U[3] = { pU[0][vj_vid],pU[1][vj_vid],pU[2][vj_vid] };
		if (vj_lid != 13) {
			for (int k = 0; k < 3; k++) {
				for (int j = 0; j < 3; j++) {
					KeU[k] += penalty * KE[k + vi * 3][j + vj * 3] * U[j];
				}
			}
		}
		if (vj_lid == 13) {
			for (int i = 0; i < 9; i++) {
				S[i] = penalty * KE[vi * 3 + i / 3][vi * 3 + i % 3];
			}
		}
	}

_blocksum:

	if (warpId >= 4) {
		for (int i = 0; i < 3; i++) {
			sumKeU[i][warpId - 4][warpTid] = KeU[i];
		}
		for (int i = 0; i < 9; i++) {
			sumS[i][warpId - 4][warpTid] = S[i];
		}
	}
	__syncthreads();

	if (warpId < 4) {
		for (int i = 0; i < 3; i++) {
			sumKeU[i][warpId][warpTid] += KeU[i];
		}
		for (int i = 0; i < 9; i++) {
			sumS[i][warpId][warpTid] += S[i];
		}
	}
	__syncthreads();

	if (warpId < 2) {
		for (int i = 0; i < 3; i++) {
			sumKeU[i][warpId][warpTid] += sumKeU[i][warpId + 2][warpTid];
		}
		for (int i = 0; i < 9; i++) {
			sumS[i][warpId][warpTid] += sumS[i][warpId + 2][warpTid];
		}
	}
	__syncthreads();

	if (warpId < 1 && !invalid_node) {
		for (int i = 0; i < 3; i++) {
			KeU[i] = sumKeU[i][0][warpTid] + sumKeU[i][1][warpTid];
		}
		for (int i = 0; i < 9; i++) {
			S[i] = sumS[i][0][warpTid] + sumS[i][1][warpTid];
		}

		double newU[3] = { pU[0][vid],pU[1][vid],pU[2][vid] };
		double(*s)[3] = reinterpret_cast<double(*)[3]>(S);
		// s[][] is row major 
		newU[0] = (gF[0][vid] - s[0][1] * newU[1] - s[0][2] * newU[2] - KeU[0]) / s[0][0];
		newU[1] = (gF[1][vid] - s[1][0] * newU[0] - s[1][2] * newU[2] - KeU[1]) / s[1][1];
		newU[2] = (gF[2][vid] - s[2][0] * newU[0] - s[2][1] * newU[1] - KeU[2]) / s[2][2];
		pU[0][vid] = newU[0]; pU[1][vid] = newU[1]; pU[2][vid] = newU[2];

	}


}

// map 32 vertices to 8 warp, each warp use specific neighbor element (density rho_i)
template<int BlockSize = 32 * 8>
__global__ void gs_relax_OTFA_WS_kernel(int nv_gs, int gs_offset, float* rholist) {
	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	__shared__ double KE[24][24];

	__shared__ double sumKeU[3][4][32];

	__shared__ double sumS[9][4][32];

	// load template matrix from constant memory to shared memory
	loadTemplateMatrix(KE);

	int warpId = threadIdx.x / 32;
	int warpTid = threadIdx.x % 32;

	double KeU[3] = { 0. };
	double S[9] = { 0. };
	double* pU[3] = { gU[0],gU[1],gU[2] };

	bool invalid_node = false;
	// the id in a gs subset
	int vid = blockIdx.x * 32 + warpTid;

	// the id in total node set
	vid += gs_offset;

	int flag = gVflag[0][vid];
	int eid;
	double penalty = 0;
	int vi = 7 - warpId;
	bool viisfix;
	int* pflags;

	invalid_node |= flag & Grid::Bitmask::mask_invalid;
	if (invalid_node) goto _blocksum;

	eid = gV2E[warpId][vid];

	if (eid != -1)
		penalty = powf(rholist[eid], power_penalty[0]);
	else
		goto _blocksum;

	if (gV2V[13][vid] == -1) {
		invalid_node = true;
		goto _blocksum;
	}

	viisfix = flag & grid::Grid::Bitmask::mask_supportnodes;

	pflags = gVflag[0];

	// compute KU and S 
	for (int vj = 0; vj < 8; vj++) {
		// vjpos = epos + vjoffset
		int vjpos[3] = {
			vj % 2 + warpId % 2,
			vj % 4 / 2 + warpId % 4 / 2,
			vj / 4 + warpId / 4
		};
		int vj_lid = vjpos[0] + vjpos[1] * 3 + vjpos[2] * 9;
		int vj_vid = gV2V[vj_lid][vid];
		if (vj_vid == -1) continue;
		double U[3] = { pU[0][vj_vid],pU[1][vj_vid],pU[2][vj_vid] };

		// deal with fixed boundary
		int vjflag = pflags[vj_vid];
		bool vjisfix = vjflag & grid::Grid::Bitmask::mask_supportnodes;

		if (vj_lid != 13 && !vjisfix) {
			for (int k = 0; k < 3; k++) {
				for (int j = 0; j < 3; j++) {
					KeU[k] += penalty * KE[k + vi * 3][j + vj * 3] * U[j];
				}
			}
		}
		if (vj_lid == 13) {
			if (!vjisfix) {
				for (int i = 0; i < 9; i++) {
					S[i] = penalty * KE[vi * 3 + i / 3][vi * 3 + i % 3];
				}
			}
			else {
				S[0] = 1; S[4] = 1; S[8] = 1;
			}
		}
	}

	if (viisfix) {
		KeU[0] = 0; KeU[1] = 0; KeU[2] = 0;
	}

_blocksum:

	if (warpId >= 4) {
		for (int i = 0; i < 3; i++) {
			sumKeU[i][warpId - 4][warpTid] = KeU[i];
		}
		for (int i = 0; i < 9; i++) {
			sumS[i][warpId - 4][warpTid] = S[i];
		}
	}
	__syncthreads();

	if (warpId < 4) {
		for (int i = 0; i < 3; i++) {
			sumKeU[i][warpId][warpTid] += KeU[i];
		}
		for (int i = 0; i < 9; i++) {
			sumS[i][warpId][warpTid] += S[i];
		}
	}
	__syncthreads();

	if (warpId < 2) {
		for (int i = 0; i < 3; i++) {
			sumKeU[i][warpId][warpTid] += sumKeU[i][warpId + 2][warpTid];
		}
		for (int i = 0; i < 9; i++) {
			sumS[i][warpId][warpTid] += sumS[i][warpId + 2][warpTid];
		}
	}
	__syncthreads();

	if (warpId < 1 && !invalid_node) {
		for (int i = 0; i < 3; i++) {
			KeU[i] = sumKeU[i][0][warpTid] + sumKeU[i][1][warpTid];
		}
		for (int i = 0; i < 9; i++) {
			S[i] = sumS[i][0][warpTid] + sumS[i][1][warpTid];
		}

		double newU[3] = { pU[0][vid],pU[1][vid],pU[2][vid] };
		double(*s)[3] = reinterpret_cast<double(*)[3]>(S);
		// s[][] is row major 
		newU[0] = (gF[0][vid] - s[0][1] * newU[1] - s[0][2] * newU[2] - KeU[0]) / s[0][0];
		newU[1] = (gF[1][vid] - s[1][0] * newU[0] - s[1][2] * newU[2] - KeU[1]) / s[1][1];
		newU[2] = (gF[2][vid] - s[2][0] * newU[0] - s[2][1] * newU[1] - KeU[2]) / s[2][2];
		pU[0][vid] = newU[0]; pU[1][vid] = newU[1]; pU[2][vid] = newU[2];

	}

}

void Grid::gs_relax(int n_times)
{
	if (is_dummy()) return;
	use_grid();
	cuda_error_check;
	if (_layer == 0) {
		for (int n = 0; n < n_times; n++) {
			int gs_offset = 0;
			for (int i = 0; i < 8; i++) {
				constexpr int BlockSize = 32 * 8;
				size_t grid_size, block_size;
				make_kernel_param(&grid_size, &block_size, gs_num[i] * 8, BlockSize);
				if (_mode == no_support_constrain_force_direction || _mode == no_support_free_force) {
					gs_relax_OTFA_NS_kernel<BlockSize> << <grid_size, block_size >> > (gs_num[i], gs_offset, _gbuf.rho_e);
				}
				else if (_mode == with_support_constrain_force_direction || _mode == with_support_free_force) {
					gs_relax_OTFA_WS_kernel<BlockSize> << <grid_size, block_size >> > (gs_num[i], gs_offset, _gbuf.rho_e);
				}
				//hipDeviceSynchronize();
				//cuda_error_check;
				gs_offset += gs_num[i];
			}
			hipDeviceSynchronize();
			cuda_error_check;
		}
	}
	else {
		check_array_len(_gbuf.rxStencil, 27 * 9 * n_gsvertices);
		for (int n = 0; n < n_times; n++) {
			int gs_offset = 0;
			for (int i = 0; i < 8; i++) {
				size_t grid_size, block_size;
				constexpr int BlockSize = 32 * 13;
				make_kernel_param(&grid_size, &block_size, gs_num[i] * 13, BlockSize);
				gs_relax_kernel<BlockSize> << <grid_size, block_size >> > (n_gsvertices, gs_num[i], _gbuf.rxStencil, gs_offset);
				//hipDeviceSynchronize();
				//cuda_error_check;
				gs_offset += gs_num[i];
			}
			hipDeviceSynchronize();
			cuda_error_check;
		}
	}
}

// map 1 vertices to 32 threads(1 warp), 4 vertices in 1 block
template<int BlockSize = 32 * 4 >
__global__ void restrict_adjoint_stencil_nondyadic_OTFA_constrain_force_direction_kernel_2(
	int nv_coarse, double* rxcoarse_, int nv_fine, float* rhofine, int* vfineflag, gBitSAT<unsigned int> vloadsat
) {
	size_t tid = size_t(blockDim.x) * blockIdx.x + threadIdx.x;
	int warpid = threadIdx.x / 32;
	int warptid = threadIdx.x % 32;

	GraftArray<double, 27, 9> rxCoarse(rxcoarse_, nv_coarse);

	__shared__ double KE[24][24];
	__shared__ double W[4][4][4];

	__shared__ double sumCoarseStencil[BlockSize / 32][27][32];

	// load template matrix from constant memory to shared memory
	loadTemplateMatrix(KE);

	// compute weight
	if (threadIdx.x < 64) {
		int i = threadIdx.x % 4;
		int j = threadIdx.x % 16 / 4;
		int k = threadIdx.x / 16;
		W[k][j][i] = (4 - i)*(4 - j)*(4 - k) / 64.f;
	}
	__syncthreads();
	
	// init coarseStencil
	//initSharedMem(&coarseStencil[0][0][0], sizeof(coarseStencil) / sizeof(double));
	double coarseStencil[27] = { 0. };

	bool validthread = true;

	int ke_id = (blockIdx.x * (BlockSize / 32) + warpid) / nv_coarse;

	// reorder K3 in row major order
	int k3row = ke_id / 3;
	int k3col = ke_id % 3;

	int vid = (blockIdx.x * (BlockSize / 32) + warpid) % nv_coarse;

	int flagword;
	float power;
	double* gvtan[2][3];

	if (ke_id >= 9) {
		validthread = false;
		goto __blocksum;
	}

	flagword = gVflag[0][vid];

	if (flagword & Grid::Bitmask::mask_invalid) {
		validthread = false;
		goto __blocksum;
	}

	power = power_penalty[0];

	for (int i = 0; i < 2; i++)
		for (int j = 0; j < 3; j++) gvtan[i][j] = gLoadtangent[i][j];


	// traverse neighbor nodes of fine element center (which is the vertex on fine fine grid)
	for (int ibase : {0, 32}) {
		int i = ibase + warptid;
		int i2[3] = { (i % 4) * 2 + 1 ,(i % 16 / 4) * 2 + 1 ,(i / 16) * 2 + 1 };
		//int m2 = i2[0] + i2[1] + i2[2] - 3;

		// get fine element center vertex
		int vn = gV2VfineC[i][vid];

		if (vn == -1) continue;

		// should traverse 7x7x7 neigbor nodes, and sum their weighted stencil, to reduce bandwidth, we traverse 8x8x8 elements 
		// traverse the neighbor fine fine element of this vertex and assembly the element matrices
		for (int j = 0; j < 8; j++) {
			int efineid = gVfine2Efine[j][vn];

			if (efineid == -1) continue;

			float rho_p = powf(rhofine[efineid], power);

			int epos[3] = { i2[0] + j % 2 - 1,i2[1] + j % 4 / 2 - 1,i2[2] + j / 4 - 1 };

			// prefecth the flag of eight vertex
			int vload[8] = { -1, -1, -1, -1, -1, -1, -1, -1 };
			for (int k = 0; k < 8; k++) {
				int vklid = j % 2 + k % 2 +
					(j / 2 % 2 + k / 2 % 2) * 3 +
					(j / 4 + k / 4) * 9;
				int vkvid = gVfine2Vfine[vklid][vn];
				int vkflag = vfineflag[vkvid];
				if (vkflag & Grid::Bitmask::mask_loadnodes) {
					vload[k] = vloadsat(vkvid);
				}
			}

			// traverse the vertex of neighbor element (rows of element matrix), compute the weight on this vertex 
			for (int ki = 0; ki < 8; ki++) {
				int vipos[3] = { epos[0] + ki % 2,epos[1] + ki % 4 / 2,epos[2] + ki / 4 };
				int wipos[3] = { abs(vipos[0] - 4),abs(vipos[1] - 4),abs(vipos[2] - 4) };
				if (wipos[0] >= 4 || wipos[1] >= 4 || wipos[2] >= 4) continue;
				double w_ki = W[wipos[0]][wipos[1]][wipos[2]] * rho_p;
				//double w_ki = (4 - wipos[0]) * (4 - wipos[1]) * (4 - wipos[2]) / 64.0;

				// fetch vi tangent vector if vi is load node
				double n1[3] = { 0. };
				n1[k3row] = 1;
				if (vload[ki] != -1) {
					n1[k3row] = 0;
					if (k3row < 2) {
						for (int m = 0; m < 3; m++) n1[m] = gvtan[k3row][m][vload[ki]];
					}
				}

				// traverse another vertex of neighbor element (cols of element matrix), get the 3x3 Ke and multiply the row weights
				for (int kj = 0; kj < 8; kj++) {
					int kjpos[3] = { epos[0] + kj % 2 , epos[1] + kj % 4 / 2 , epos[2] + kj / 4 };
					double wk = 0;

					double n2[3] = { 0. };
					n2[k3col] = 1;
					// check whether vi and vj are load nodes, multiply N if they are 
					if (vload[kj] != -1) {
						n2[k3col] = 0;
						if (k3col < 2) {
							for (int m = 0; m < 3; m++) n2[m] = gvtan[k3col][m][vload[kj]];
						}
					}

					// compute N * K * N^T
					for (int m = 0; m < 3; m++) {
						for (int n = 0; n < 3; n++) {
							wk += n1[m] * KE[ki * 3 + m][kj * 3 + n] * n2[n];
						}
					}

					// set degenerated diagonal as 1 
					if (ki == kj && vload[ki] != -1 && k3row == 2 && k3col == 2) {
						wk = 1;
					}

					// multiply weight on ki
					wk *= w_ki;

				_splitwk:
					//  the weighted element matrix should split to coarse vertex, traverse the coarse vertices and split 3x3 Ke to coarse vertex by splitting weights
					for (int vsplit = 0; vsplit < 27; vsplit++) {
						int vsplitpos[3] = { vsplit % 3 * 4, vsplit % 9 / 3 * 4,vsplit / 9 * 4 };
						int wjpos[3] = { abs(vsplitpos[0] - kjpos[0]), abs(vsplitpos[1] - kjpos[1]), abs(vsplitpos[2] - kjpos[2]) };
						if (wjpos[0] >= 4 || wjpos[1] >= 4 || wjpos[2] >= 4) continue;
						double wkw = wk * W[wjpos[0]][wjpos[1]][wjpos[2]];
						//double wkw = wk * (4 - wjpos[0]) * (4 - wjpos[1]) * (4 - wjpos[2]) / 64.;
						coarseStencil[vsplit]/*[warpid][warptid]*/ += wkw;
					}
				}
			}
		}
	}

__blocksum:

	for (int i = 0; i < 27; i++) {
		sumCoarseStencil[warpid][i][warptid] = coarseStencil[i];
	}

	__syncthreads();

#if 1
	// warp reduce sum on sumCoarseStencil[][][*]
	if (warptid < 16) {
		for (int i = 0; i < 27; i++) {
			sumCoarseStencil[warpid][i][warptid] += sumCoarseStencil[warpid][i][warptid + 16];
		}
	}
	if (warptid < 8) {
		for (int i = 0; i < 27; i++) {
			sumCoarseStencil[warpid][i][warptid] += sumCoarseStencil[warpid][i][warptid + 8];
		}
	}
	if (warptid < 4) {
		for (int i = 0; i < 27; i++) {
			sumCoarseStencil[warpid][i][warptid] += sumCoarseStencil[warpid][i][warptid + 4];
		}
	}
	if (warptid < 2) {
		for (int i = 0; i < 27; i++) {
			sumCoarseStencil[warpid][i][warptid] += sumCoarseStencil[warpid][i][warptid + 2];
		}
	}
	if (warptid < 1 && validthread) {
		for (int i = 0; i < 27; i++) {
			rxCoarse[i][ke_id][vid] = sumCoarseStencil[warpid][i][0] + sumCoarseStencil[warpid][i][1];
		}
	}
#else

	if (warptid == 0 && validthread) {
		for (int i = 0; i < 27; i++) {
			double sumst = 0;
			for (int j = 0; j < 32; j++) {
				sumst += sumCoarseStencil[warpid][i][j];
			}
			rxCoarse[i][ke_id][vid] = sumst;
		}
	}

#endif
}

__global__ void update_residual_kernel(int nv, double* rxstencil) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= nv) return;
	int vid = tid;

	GraftArray<double, 27, 9> stencil(rxstencil, nv);
	//double f[3] = { gF[0][vid],gF[1][vid],gF[2][vid] };
	double KU[3] = { 0. };
	for (int i = 0; i < 27; i++) {
		int vj = gV2V[i][vid];
		if (vj == -1) continue;
		double u[3] = { gU[0][vj],gU[1][vj],gU[2][vj] };
		for (int row = 0; row < 3; row++) {
			for (int col = 0; col < 3; col++) {
				KU[row] += stencil[i][row * 3 + col][vid] * u[col];
			}
		}
	}

	for (int i = 0; i < 3; i++) {
		gR[i][vid] = gF[i][vid] - KU[i];
	}
}

// map 32 vertices to 13 warp
template<int SetBlockSize = 32 * 13>
__global__ void update_residual_kernel_1(int nv, double* rxstencil) {
	GraftArray<double, 27, 9> stencil(rxstencil, nv);
	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	//int mode = gmode[0];

	__shared__ double sumAu[3][13][32];
	int warpId = threadIdx.x / 32;
	int warpTid = threadIdx.x % 32;
	int vid = blockIdx.x * 32 + warpTid;

	double Au[3] = { 0.f,0.f,0.f };

	int flag;

	bool invalid_node = true;

	if (vid < nv) {
		flag = gVflag[0][vid];
		invalid_node = flag & Grid::Bitmask::mask_invalid;
		if (invalid_node) goto _blockSum;
		for (auto i : { 0,14 }) {
			double displacement[3];
			int neigh_th = warpId + i;
			int neigh = gV2V[neigh_th][vid];
			if (neigh == -1) continue;

			for (int j = 0; j < 3; j++) displacement[j] = gU[j][neigh];

			// K3 is ordered in row major 
			// traverse rows 
			for (int j = 0; j < 3; j++) {
				int jrows = j * 3;
				// traverse columns, dot u 
				for (int k = 0; k < 3; k++) {
					Au[j] += stencil[neigh_th][jrows + k][vid] * displacement[k];
				}
			}

		}
	}

_blockSum:

	for (int i = 0; i < 3; i++) {
		sumAu[i][warpId][warpTid] = Au[i];
	}
	__syncthreads();

	// gather all part
	if (warpId < 7) {
		for (int i = 0; i < 3; i++) {
			int addId = warpId + 7;
			if (addId < 13) {
				sumAu[i][warpId][warpTid] += sumAu[i][addId][warpTid];
			}
		}
	}
	__syncthreads();
	if (warpId < 4) {
		for (int i = 0; i < 3; i++) {
			int addId = warpId + 4;
			if (addId < 7) {
				sumAu[i][warpId][warpTid] += sumAu[i][addId][warpTid];
			}
		}
	}
	__syncthreads();
	if (warpId < 2) {
		for (int i = 0; i < 3; i++) {
			int addId = warpId + 2;
			sumAu[i][warpId][warpTid] += sumAu[i][addId][warpTid];
		}
	}
	__syncthreads();
	if (warpId < 1) {
		for (int i = 0; i < 3; i++) {
			int addId = warpId + 1;
			Au[i] = sumAu[i][warpId][warpTid] + sumAu[i][addId][warpTid];
		}
	}
	//__syncthreads();

	if (vid < nv) {
		double displacement[3] = { 0. };
		if (warpId == 0) {
			for (int i = 0; i < 3; i++) displacement[i] = gU[i][vid];
			for (int i = 0; i < 3; i++) {
				Au[0] += stencil[13][i][vid] * displacement[i];
				Au[1] += stencil[13][3 + i][vid] * displacement[i];
				Au[2] += stencil[13][6 + i][vid] * displacement[i];
			}
			for (int i = 0; i < 3; i++) {
				gR[i][vid] = gF[i][vid] - Au[i];
			}
		}
	}
}

__global__ void update_residual_OTFA_NS_kernel(int nv, float* rholist) {

	__shared__ double KE[24][24];

	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	loadTemplateMatrix(KE);

	if (tid >= nv) return;

	int vid = tid;

	//int mode = gmode[0];
	int v2v[27];
	loadNeighborNodes(vid, v2v);

	double KU[3] = { 0.,0.,0. };
	float power = power_penalty[0];
	for (int i = 0; i < 8; i++) {
		int eid = gV2E[i][vid];
		if (eid == -1) continue;
		double penalty = powf(rholist[eid], power);
		int vi = 7 - i;
		for (int vj = 0; vj < 8; vj++) {
			int vjpos[3] = {
				vj % 2 + i % 2,
				vj % 4 / 2 + i % 4 / 2,
				vj / 4 + i / 4
			};
			int vj_lid = vjpos[0] + vjpos[1] * 3 + vjpos[2] * 9;
			int vj_vid = v2v[vj_lid];
			if (vj_vid == -1) {
				// DEBUG
				printf("-- error in update residual otfa\n");
				continue;
			}
			double u[3] = { gU[0][vj_vid],gU[1][vj_vid],gU[2][vj_vid] };
			for (int row = 0; row < 3; row++) {
				for (int col = 0; col < 3; col++) {
					KU[row] += penalty * KE[row + vi * 3][col + vj * 3] * u[col];
				}
			}
		}

	}

	for (int i = 0; i < 3; i++) {
		gR[i][vid] = gF[i][vid] - KU[i];
	}
}

__global__ void update_residual_OTFA_WS_kernel(int nv, float* rholist) {

	__shared__ double KE[24][24];

	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	loadTemplateMatrix(KE);

	if (tid >= nv) return;

	int vid = tid;

	// add fixed flag check
	bool vfix[27], vload[27];
	int v2v[27];
	loadNeighborNodesAndFlags(vid, v2v, vfix, vload);

	double KU[3] = { 0.,0.,0. };
	float power = power_penalty[0];
	for (int i = 0; i < 8; i++) {
		int eid = gV2E[i][vid];
		if (eid == -1) continue;
		double penalty = powf(rholist[eid], power);
		int vi = 7 - i;
		for (int vj = 0; vj < 8; vj++) {
			int vjpos[3] = {
				vj % 2 + i % 2,
				vj % 4 / 2 + i % 4 / 2,
				vj / 4 + i / 4
			};
			int vj_lid = vjpos[0] + vjpos[1] * 3 + vjpos[2] * 9;
			int vj_vid = v2v[vj_lid];
			if (vj_vid == -1) {
				// DEBUG
				printf("-- error in update residual otfa\n");
				continue;
			}
			double u[3] = { gU[0][vj_vid],gU[1][vj_vid],gU[2][vj_vid] };
			if (vfix[vj_lid]) {
				u[0] = 0; u[1] = 0; u[2] = 0;
			}
			for (int row = 0; row < 3; row++) {
				for (int col = 0; col < 3; col++) {
					KU[row] += penalty * KE[row + vi * 3][col + vj * 3] * u[col];
				}
			}
		}
	}

	if (vfix[13]) {
		KU[0] = 0; KU[1] = 0; KU[2] = 0;
	}

	for (int i = 0; i < 3; i++) {
		gR[i][vid] = gF[i][vid] - KU[i];
	}
}

template<int SetBlockSize = 32 * 8>
__global__ void update_residual_OTFA_WS_kernel_1(int nv, float* rholist) {

	__shared__ double KE[24][24];
	__shared__ double sumKeU[3][4][32];

	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	loadTemplateMatrix(KE);

	int warpId = threadIdx.x / 32;
	int warpTid = threadIdx.x % 32;

	double KeU[3] = { 0.,0.,0. };

	float power = power_penalty[0];

	int vid = blockIdx.x * 32 + warpTid;

	// add fixed flag check
	bool vfix[27], vload[27];
	int v2v[27];

	if (vid >= nv) goto __blocksum;

	loadNeighborNodesAndFlags(vid, v2v, vfix, vload);

	// sum a element
	{
		int i = warpId;
		int eid = gV2E[i][vid];
		double penalty;
		int vi = 7 - i;
		if (eid == -1) goto __blocksum;
		penalty = powf(rholist[eid], power);
		for (int vj = 0; vj < 8; vj++) {
			int vjpos[3] = {
				vj % 2 + i % 2,
				vj % 4 / 2 + i % 4 / 2,
				vj / 4 + i / 4
			};
			int vj_lid = vjpos[0] + vjpos[1] * 3 + vjpos[2] * 9;
			int vj_vid = v2v[vj_lid];
			if (vj_vid == -1) {
				// DEBUG
				printf("-- error in update residual otfa\n");
				continue;
			}
			double u[3] = { gU[0][vj_vid],gU[1][vj_vid],gU[2][vj_vid] };
			if (vfix[vj_lid]) {
				u[0] = 0; u[1] = 0; u[2] = 0;
			}
			for (int row = 0; row < 3; row++) {
				for (int col = 0; col < 3; col++) {
					KeU[row] += penalty * KE[row + vi * 3][col + vj * 3] * u[col];
				}
			}
		}
	}

__blocksum:
	if (warpId >= 4) {
		for (int i = 0; i < 3; i++) { sumKeU[i][warpId - 4][warpTid] = KeU[i]; }
	}
	__syncthreads();

	if (warpId < 4) {
		for (int i = 0; i < 3; i++) { sumKeU[i][warpId][warpTid] += KeU[i]; }
	}
	__syncthreads();

	if (warpId < 2) {
		for (int i = 0; i < 3; i++) { sumKeU[i][warpId][warpTid] += sumKeU[i][warpId + 2][warpTid]; }
	}
	__syncthreads();

	if (warpId < 1 && v2v[13] != -1) {
		for (int i = 0; i < 3; i++) { KeU[i] = sumKeU[i][0][warpTid] + sumKeU[i][1][warpTid]; }

		if (vfix[13]) { KeU[0] = 0; KeU[1] = 0; KeU[2] = 0; }
		
		for (int i = 0; i < 3; i++) { gR[i][vid] = gF[i][vid] - KeU[i]; }
	}
}

void Grid::update_residual(void)
{
	if (is_dummy()) return;
	use_grid();
	size_t grid_size, block_size;
	if (_layer == 0) {
		if (_mode == no_support_constrain_force_direction || _mode == no_support_free_force) {
			make_kernel_param(&grid_size, &block_size, n_gsvertices, 512);
			update_residual_OTFA_NS_kernel << <grid_size, block_size >> > (n_gsvertices, _gbuf.rho_e);
		}
		else if (_mode == with_support_constrain_force_direction || _mode == with_support_free_force) {
#if 1
			make_kernel_param(&grid_size, &block_size, n_gsvertices, 256);
			update_residual_OTFA_WS_kernel << <grid_size, block_size >> > (n_gsvertices, _gbuf.rho_e);
#else
			make_kernel_param(&grid_size, &block_size, n_gsvertices * 8, 32 * 8);
			update_residual_OTFA_WS_kernel_1 << <grid_size, block_size >> > (n_gsvertices, _gbuf.rho_e);
#endif
		}
		hipDeviceSynchronize();
		cuda_error_check;
	}
	else {
#if 0
		make_kernel_param(&grid_size, &block_size, n_gsvertices, 512);
		update_residual_kernel << <grid_size, block_size >> > (n_gsvertices, _gbuf.rxStencil);
#else
		make_kernel_param(&grid_size, &block_size, n_gsvertices * 13, 32 * 13);
		update_residual_kernel_1 << <grid_size, block_size >> > (n_gsvertices, _gbuf.rxStencil);

#endif
		hipDeviceSynchronize();
		cuda_error_check;
	}
}

__global__ void restrict_residual_kernel(int nv) {
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if (tid >= nv) return;
	
	//int mode = gmode[0];

	double res[3] = { 0.f };

	// volume center
	{
		int neigh = gV2Vfine[13][tid];
		if (neigh != -1) {
			for (int i = 0; i < 3; i++) {
				res[i] += gRfine[i][neigh];
			}
		}
	}

	// volume vertex
	for (int j : {0, 2, 6, 8, 18, 20, 24, 26}) {
		int neigh = gV2Vfine[j][tid];
		if (neigh != -1) {
			for (int i = 0; i < 3; i++) {
				res[i] += gRfine[i][neigh] * (1.0f / 8);
			}
		}
	}
	// face center
	for (int j : {4, 10, 12, 14, 16, 22}) {
		int neigh = gV2Vfine[j][tid];
		if (neigh != -1) {
			for (int i = 0; i < 3; i++) {
				res[i] += gRfine[i][neigh] * (1.0f / 2);
			}
		}
	}
	// edge center
	for (int j : {1, 3, 5, 7, 9, 11, 15, 17, 19, 21, 23, 25}) {
		int neigh = gV2Vfine[j][tid];
		if (neigh != -1) {
			for (int i = 0; i < 3; i++) {
				res[i] += gRfine[i][neigh] * (1.0f / 4);
			}
		}
	}

__writeResidual:

	for (int i = 0; i < 3; i++) {
		gF[i][tid] = res[i] /*/ 8*/;
	}
}

__global__ void restrict_residual_nondyadic_kernel(int nv) {
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	//if (tid >= nv) return;

	//int mode = gmode[0];
	__shared__ double W[4][4][4];
	__shared__ int* vfine2vfine[27];

	if (threadIdx.x < 64) {
		int k = threadIdx.x % 4;
		int j = threadIdx.x / 4 % 4;
		int i = threadIdx.x / 16;
		W[i][j][k] = ((4 - i)*(4 - j)*(4 - k)) / 64.0;
		if (threadIdx.x < 27) {
			vfine2vfine[threadIdx.x] = gVfine2Vfine[threadIdx.x];
		}
	}
	__syncthreads();

	if (tid >= nv) return;

	int vid = tid;

	int aFlag[(7 * 7 * 7) / (sizeof(int) * 8) + 1] = { 0 };

	double sumR[3] = { 0. };

	// DEBUG
	//if (sumR[0] != 0 || sumR[1] != 0 || sumR[2] != 0) { printf("\033[31m-- kernel error, nonzero init at file %s, line %d\033[0m\n", __FILE__, __LINE__); }

	double* rfine[3] = { gRfine[0], gRfine[1], gRfine[2] };

	for (int i = 0; i < 64; i++) {
		int vff = gV2VfineC[i][vid];
		if (vff == -1) continue;
		int basepos[3] = { i % 4 * 2 - 3,i % 16 / 4 * 2 - 3,i / 16 * 2 - 3 };
		for (int dx = -1; dx <= 1; dx++) {
			int xj = basepos[0] + dx;
			if (xj <= -4 || xj >= 4) continue;
			for (int dy = -1; dy <= 1; dy++) {
				int yj = basepos[1] + dy;
				if (yj <= -4 || yj >= 4) continue;
				for (int dz = -1; dz <= 1; dz++) {
					int zj = basepos[2] + dz;
					if (zj <= -4 || zj >= 4) continue;
					int jid = xj + 3 + (yj + 3) * 7 + (zj + 3) * 49;
					if (read_gbit(aFlag, jid)) continue;
					set_gbit(aFlag, jid);
					int djid = (dx + 1) + (dy + 1) * 3 + (dz + 1) * 9;
					int vj_vid = vfine2vfine[djid][vff];
					if (vj_vid == -1) continue;
					double r[3] = { rfine[0][vj_vid], rfine[1][vj_vid], rfine[2][vj_vid] };
					//double weight = (4 - abs(xj))*(4 - abs(yj))*(4 - abs(zj)) / 64.0;
					double weight = W[abs(xj)][abs(yj)][abs(zj)];
					for (int k = 0; k < 3; k++) sumR[k] += weight * r[k];
				}
			}
		}
	}

	for (int k = 0; k < 3; k++) { gF[k][vid] = sumR[k]; }
}

void Grid::restrict_residual(void)
{
	use_grid();

	size_t grid_size, block_size;
	if (_layer == 2 && is_skip()) {
		make_kernel_param(&grid_size, &block_size, n_gsvertices, 256);
		restrict_residual_nondyadic_kernel << <grid_size, block_size >> > (n_gsvertices);
		hipDeviceSynchronize();
		cuda_error_check;
	}
	else if (_layer == 0) {
		msg() << "\033[31mCannot restrict residual to finest layer" << "\033[0m" << std::endl;
	}
	else {
		make_kernel_param(&grid_size, &block_size, n_gsvertices, 512);
		restrict_residual_kernel << <grid_size, block_size >> > (n_gsvertices);
		hipDeviceSynchronize();
		cuda_error_check;
	}
}

__global__ void prolongate_correction_kernel(int nv) {
	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	//int mode = gmode[0];

	if (tid >= nv) return;

	int vid = tid;


	double c[3] = { 0. };

	double* pU[3] = { gUcoarse[0],gUcoarse[1],gUcoarse[2] };

	int flag = gVflag[0][vid];
	if (flag& Grid::Bitmask::mask_invalid) return;

	int posInE[3] = {
		((flag & Grid::Bitmask::mask_xmod7) >> Grid::Bitmask::offset_xmod7) % 2,
		((flag & Grid::Bitmask::mask_ymod7) >> Grid::Bitmask::offset_ymod7) % 2,
		((flag & Grid::Bitmask::mask_zmod7) >> Grid::Bitmask::offset_zmod7) % 2
	};

	for (int i = 0; i < 8; i++) {
		int vcoarsepos[3] = { i % 2 * 2, i % 4 / 2 * 2, i / 4 * 2 };
		int wpos[3] = { abs(vcoarsepos[0] - posInE[0]), abs(vcoarsepos[1] - posInE[1]), abs(vcoarsepos[2] - posInE[2]) };
		if (wpos[0] >= 2 || wpos[1] >= 2 || wpos[2] >= 2) continue;
		double weight = (2 - wpos[0]) * (2 - wpos[1]) * (2 - wpos[2]) / 8.;
		int vcoarseid = gV2Vcoarse[i][vid];
		if (vcoarseid == -1) continue;
		for (int j = 0; j < 3; j++) {
			c[j] += weight * pU[j][vcoarseid];
		}
	}

	for (int i = 0; i < 3; i++) {
		gU[i][vid] += c[i];
	}
}


__global__ void prolongate_correction_nondyadic_kernel(int nv, int* vbitflag) {
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if (tid >= nv) return;

	int vid = tid;

	//int mode = gmode[0];

	double c[3] = { 0. };

	int flagword = vbitflag[vid];

	if (flagword & Grid::Bitmask::mask_invalid) return;

	int posInE[3] = {
		(flagword & Grid::Bitmask::mask_xmod7) >> Grid::Bitmask::offset_xmod7,
		(flagword & Grid::Bitmask::mask_ymod7) >> Grid::Bitmask::offset_ymod7,
		(flagword & Grid::Bitmask::mask_zmod7) >> Grid::Bitmask::offset_zmod7
	};
	for (int i = 0; i < 3; i++) posInE[i] %= 4;

	int nei_counter = 0;
	// traverse vertex of coarse element which contains this fine vertex
	for (int i = 0; i < 8; i++) {
		int finepos[3] = { i % 2 * 4, i % 4 / 2 * 4, i / 4 * 4 };
		int wpos[3] = { abs(finepos[0] - posInE[0]), abs(finepos[1] - posInE[1]), abs(finepos[2] - posInE[2]) };
		if (wpos[0] >= 4 || wpos[1] >= 4 || wpos[2] >= 4) continue;
		double weight = (4 - wpos[0]) * (4 - wpos[1]) * (4 - wpos[2]) / 64.0;
		int coarseid = gV2Vcoarse[i][vid];
		if (coarseid == -1) continue;
		for (int j = 0; j < 3; j++) c[j] += weight * gUcoarse[j][coarseid];
	}

	for (int i = 0; i < 3; i++) {
		gU[i][vid] += c[i];
	}
}


void Grid::prolongate_correction(void)
{
	if (is_dummy()) return;
	use_grid();
	size_t grid_size, block_size;
	if (_layer == 0 && is_skip()) {
		make_kernel_param(&grid_size, &block_size, n_gsvertices, 512);
		prolongate_correction_nondyadic_kernel << <grid_size, block_size >> > (n_gsvertices, _gbuf.vBitflag);
		hipDeviceSynchronize();
		cuda_error_check;
	}
	else {
		make_kernel_param(&grid_size, &block_size, n_gsvertices, 512);
		prolongate_correction_kernel << <grid_size, block_size >> > (n_gsvertices);
		hipDeviceSynchronize();
		cuda_error_check;
	}
}

void Grid::reset_displacement(void)
{
	for (int i = 0; i < 3; i++) {
		init_array(_gbuf.U[i], 0., n_gsvertices);
	}
}

void Grid::reset_force(void)
{
	cuda_error_check;
	for (int i = 0; i < 3; i++) {
		init_array(_gbuf.F[i], 0., n_gsvertices);
	}
}

void Grid::reset_residual(void)
{
	for (int i = 0; i < 3; i++) {
		init_array(_gbuf.R[i], 0., n_gsvertices);
	}
}

double Grid::v3norm(double* v[3])
{
	double s = norm(v[0], v[1], v[2], n_nodes());
	return s;
}

double Grid::relative_residual(void)
{
	double r = v3norm(_gbuf.R);
	double f = v3norm(_gbuf.F);
	return r / f;
}

double Grid::residual(void)
{
	return v3norm(_gbuf.R);
}


//__global__ void mark_surface_nodes_kernel(int nv, int* vflag, int* eflag) {
//	int tid = threadIdx.x + blockDim.x*blockIdx.x;
//	if (tid >= nv) return;
//
//	int vid = tid;
//
//	bool surf = false;
//
//	bool axisHasNeighbor[3] = { false, false, false };
//
//	bool solid_flag[2][2][2];
//	for (int i = 0; i < 8; i++) {
//		int eid = gV2E[i][vid];
//		solid_flag[i % 2][i % 4 / 2][i / 4] = (eid != -1);
//	}
//
//	for (int i = 0; i < 2; i++) {
//		for (int j = 0; j < 2; j++) {
//			axisHasNeighbor[0] |= solid_flag[0][i][j] && solid_flag[1][i][j];
//			axisHasNeighbor[1] |= solid_flag[i][0][j] && solid_flag[i][1][j];
//			axisHasNeighbor[2] |= solid_flag[i][j][0] && solid_flag[i][j][1];
//		}
//	}
//
//	surf = (!axisHasNeighbor[0]) || (!axisHasNeighbor[1]) || (!axisHasNeighbor[2]);
//
//	int word = vflag[vid];
//	if (surf) {
//		word |= Grid::Bitmask::mask_surfacenodes;
//	}
//	else {
//		word &= ~(int)Grid::Bitmask::mask_surfacenodes;
//	}
//	vflag[vid] = word;
//}

//void Grid::mark_surface_nodes_g(void)
//{
//	use_grid();
//	size_t grid_size, block_size;
//	make_kernel_param(&grid_size, &block_size, n_gsvertices, 512);
//	mark_surface_nodes_kernel << <grid_size, block_size >> > (n_gsvertices, _gbuf.vBitflag, _gbuf.eBitflag);
//	hipDeviceSynchronize();
//	cuda_error_check;
//}

__global__ void mark_surface_nodes_kernel(int nv, devArray_t<int*, 8> v2elist, int* vflag) {
	int tid = threadIdx.x + blockDim.x*blockIdx.x;
	if (tid >= nv) return;

	int vid = tid;

	bool surf = false;

	bool axisHasNeighbor[3] = { false, false, false };

	bool solid_flag[2][2][2];
	for (int i = 0; i < 8; i++) {
		int eid = v2elist[i][vid];
		solid_flag[i % 2][i % 4 / 2][i / 4] = (eid != -1);
	}

	for (int i = 0; i < 2; i++) {
		for (int j = 0; j < 2; j++) {
			axisHasNeighbor[0] |= solid_flag[0][i][j] && solid_flag[1][i][j];
			axisHasNeighbor[1] |= solid_flag[i][0][j] && solid_flag[i][1][j];
			axisHasNeighbor[2] |= solid_flag[i][j][0] && solid_flag[i][j][1];
		}
	}

	surf = (!axisHasNeighbor[0]) || (!axisHasNeighbor[1]) || (!axisHasNeighbor[2]);

	int word = vflag[vid];
	if (surf) {
		word |= Grid::Bitmask::mask_surfacenodes;
	}
	else {
		word &= ~(int)Grid::Bitmask::mask_surfacenodes;
	}
	vflag[vid] = word;
}

void Grid::mark_surface_nodes_g(int nv, int* v2e[8], int* vflag)
{
	devArray_t<int*, 8> v2elist;
	for (int i = 0; i < 8; i++) v2elist[i] = v2e[i];

	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, nv, 512);
	mark_surface_nodes_kernel << <grid_size, block_size >> > (nv, v2elist, _gbuf.vBitflag);
	hipDeviceSynchronize();
	cuda_error_check;
}

__global__ void mark_surface_elements_kernel(int nv, devArray_t<int*, 8> v2elist, int* vflag, int* eflag) {
	size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= nv) return;

	int vfw = vflag[tid];

	if (vfw & Grid::Bitmask::mask_surfacenodes) {
		for (int i = 0; i < 8; i++) {
			int eid = v2elist[i][tid];
			if (eid == -1) continue;
			atomic_set_gbit(eflag, sizeof(int) * 8 * eid + Grid::Bitmask::offset_surfaceelements);
		}
	}

}

void grid::Grid::mark_surface_elements_g(int nv, int ne, int* v2e[8], int* vflag, int* eflag)
{
	devArray_t<int*, 8> v2elist;
	for (int i = 0; i < 8; i++) v2elist[i] = v2e[i];

	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, nv, 512);
	mark_surface_elements_kernel << <grid_size, block_size >> > (nv, v2elist, vflag, eflag);
	hipDeviceSynchronize();
	cuda_error_check;
}



std::vector<int> Grid::getVflags(void)
{
	std::vector<int> hostflag(n_gsvertices);
	hipMemcpy(hostflag.data(), _gbuf.vBitflag, sizeof(int)*n_gsvertices, hipMemcpyDeviceToHost);
	cuda_error_check;
	return hostflag;
}

std::vector<int> Grid::getEflags(void)
{
	std::vector<int> hostflag(n_gselements);
	hipMemcpy(hostflag.data(), _gbuf.eBitflag, sizeof(int)*n_gselements, hipMemcpyDeviceToHost);
	cuda_error_check;
	return hostflag;
}

void Grid::getVflags(int nv, int* dst)
{
	hipMemcpy(dst, _gbuf.vBitflag, sizeof(int)* nv, hipMemcpyDeviceToHost);
}

void Grid::setVflags(int nv, int *src)
{
	hipMemcpy(_gbuf.vBitflag, src, sizeof(int)* nv, hipMemcpyHostToDevice);
}

void Grid::getEflags(int nv, int* dst)
{
	hipMemcpy(dst, _gbuf.eBitflag, sizeof(int) * nv, hipMemcpyDeviceToHost);
}


void Grid::v3_init(double* v[3], double val[3])
{
	for (int i = 0; i < 3; i++) {
		init_array(v[i], val[i], n_gsvertices);
	}
}


void Grid::v3_minus(double* a[3], double alpha, double* b[3])
{
	double* ax = a[0], *ay = a[1], *az = a[2];
	double* bx = b[0], *by = b[1], *bz = b[2];
	size_t grid_dim, block_dim;
	make_kernel_param(&grid_dim, &block_dim, n_gsvertices, 512);
	map << <grid_dim, block_dim >> > (n_nodes(), [=]__device__(int tid) { ax[tid] -= alpha * bx[tid]; });
	hipDeviceSynchronize();
	cuda_error_check;

	map << <grid_dim, block_dim >> > (n_nodes(), [=]__device__(int tid) { ay[tid] -= alpha * by[tid]; });
	hipDeviceSynchronize();
	cuda_error_check;

	map << <grid_dim, block_dim >> > (n_nodes(), [=]__device__(int tid) { az[tid] -= alpha * bz[tid]; });
	hipDeviceSynchronize();
	cuda_error_check;
}

void Grid::v3_minus(double* dst[3], double* a[3], double alpha, double* b[3])
{
	double* ax = a[0], *ay = a[1], *az = a[2];
	double* bx = b[0], *by = b[1], *bz = b[2];
	double* dstx = dst[0], *dsty = dst[1], *dstz = dst[2];
	size_t grid_dim, block_dim;
	make_kernel_param(&grid_dim, &block_dim, n_gsvertices, 512);
	map << <grid_dim, block_dim >> > (n_nodes(), [=]__device__(int tid) { dstx[tid] = ax[tid] - alpha * bx[tid]; });
	hipDeviceSynchronize();
	cuda_error_check;

	map << <grid_dim, block_dim >> > (n_nodes(), [=]__device__(int tid) { dsty[tid] = ay[tid] - alpha * by[tid]; });
	hipDeviceSynchronize();
	cuda_error_check;

	map << <grid_dim, block_dim >> > (n_nodes(), [=]__device__(int tid) { dstz[tid] = az[tid] - alpha * bz[tid]; });
	hipDeviceSynchronize();
	cuda_error_check;
}


void Grid::v3_add(double* a[3], double alpha, double* b[3])
{
	double* ax = a[0], *ay = a[1], *az = a[2];
	double* bx = b[0], *by = b[1], *bz = b[2];
	size_t grid_dim, block_dim;
	make_kernel_param(&grid_dim, &block_dim, n_gsvertices, 512);
	map << <grid_dim, block_dim >> > (n_nodes(), [=]__device__(int tid) { ax[tid] += alpha * bx[tid]; });
	hipDeviceSynchronize();
	cuda_error_check;

	map << <grid_dim, block_dim >> > (n_nodes(), [=]__device__(int tid) { ay[tid] += alpha * by[tid]; });
	hipDeviceSynchronize();
	cuda_error_check;

	map << <grid_dim, block_dim >> > (n_nodes(), [=]__device__(int tid) { az[tid] += alpha * bz[tid]; });
	hipDeviceSynchronize();
	cuda_error_check;
}

void Grid::v3_add(double alpha, double* a[3], double beta, double* b[3])
{
	double* ax = a[0], *ay = a[1], *az = a[2];
	double* bx = b[0], *by = b[1], *bz = b[2];
	size_t grid_dim, block_dim;
	make_kernel_param(&grid_dim, &block_dim, n_gsvertices, 512);
	map << <grid_dim, block_dim >> > (n_nodes(), [=]__device__(int tid) { ax[tid] = alpha * ax[tid] + beta * bx[tid]; });
	hipDeviceSynchronize();
	cuda_error_check;

	map << <grid_dim, block_dim >> > (n_nodes(), [=]__device__(int tid) { ay[tid] = alpha * ay[tid] + beta * by[tid]; });
	hipDeviceSynchronize();
	cuda_error_check;

	map << <grid_dim, block_dim >> > (n_nodes(), [=]__device__(int tid) { az[tid] = alpha * az[tid] + beta * bz[tid]; });
	hipDeviceSynchronize();
	cuda_error_check;
}

double Grid::v3_dot(double* v[3], double* u[3])
{
	double* tmp = (double*)getTempBuf(n_gsvertices / 100 * sizeof(double));
	double s = dot(v[0], v[1], v[2], u[0], u[1], u[2], tmp, n_gsvertices);
	return s;
}

double Grid::v3_diffdot(double* v1[3], double* v2[3], double* v3[3], double* v4[3])
{
	double sum = parallel_diffdot(n_nodes(), v1, v2, v3, v4);
	cuda_error_check;
	return sum;
}

double Grid::v3_norm(double* v[3])
{
	double s = norm(v[0], v[1], v[2], n_nodes());
	return s;
}

double grid::Grid::v3_normalize(double* v[3])
{
	double nr = v3_norm(v);
	v3_scale(v, 1.0 / nr);
	return nr;
}

void grid::Grid::v3_destroy(double* dstv[3])
{
	for (int i = 0; i < 3; i++) {
		hipFree(dstv[i]);
	}
	cuda_error_check;
}

void Grid::v3_rand(double* v[3], double low, double upp)
{
	randArray(v, 3, n_gsvertices, low, upp);
}

void Grid::randForce(void)
{
	v3_rand(_gbuf.F, -1, 1);
}

double Grid::unitizeForce(void)
{
	double fnorm = v3_norm(_gbuf.F);
	//printf("-- untize f norm = %lf\n", fnorm);
	v3_scale(_gbuf.F, 1.0 / fnorm);
	return fnorm;
}

double Grid::supportForceCh(void)
{
	double * fs[4];
	getTempBufArray(fs, 4, n_loadnodes());

	getForceSupport(_gbuf.F, fs);

	double sum = parallel_diffdot(n_loadnodes(), _gbuf.Fsupport, fs, _gbuf.Fsupport, fs);
	cuda_error_check;

	return sqrt(sum);
}

double grid::Grid::supportForceCh(double* newf[3])
{
	double* newfs[4];
	getTempBufArray(newfs, 4, n_loadnodes());

	getForceSupport(newf, newfs);

	double sum = parallel_diffdot(n_loadnodes(), _gbuf.Fsupport, newfs, _gbuf.Fsupport, newfs);
	cuda_error_check;

	return sqrt(sum);
}

double Grid::supportForceNorm(void)
{
	double sum = norm(_gbuf.Fsupport[0], _gbuf.Fsupport[1], _gbuf.Fsupport[2], n_loadnodes());
	cuda_error_check;
	return sum;
}

void Grid::v3_scale(double* v[3], double ampl)
{
	double *vx = v[0], *vy = v[1], *vz = v[2];
	size_t grid_dim, block_dim;
	make_kernel_param(&grid_dim, &block_dim, n_nodes(), 512);
	map << <grid_dim, block_dim >> > (n_nodes(), [=]__device__(int tid) { vx[tid] *= ampl; });
	hipDeviceSynchronize();
	cuda_error_check;

	map << <grid_dim, block_dim >> > (n_nodes(), [=]__device__(int tid) { vy[tid] *= ampl; });
	hipDeviceSynchronize();
	cuda_error_check;

	map << <grid_dim, block_dim >> > (n_nodes(), [=]__device__(int tid) { vz[tid] *= ampl; });
	hipDeviceSynchronize();
	cuda_error_check;
}


void Grid::v3_copy(double* vsrc[3], double* vdst[3])
{
	for (int i = 0; i < 3; i++) {
		hipMemcpy(vdst[i], vsrc[i], sizeof(double)*n_nodes(), hipMemcpyDeviceToDevice);
		cuda_error_check;
	}
}

void HierarchyGrid::setMode(Mode mode)
{
	int modeid = mode;
	_mode = mode;
	Grid::_mode = mode;
	hipMemcpyToSymbol(HIP_SYMBOL(gmode), &modeid, sizeof(int));
}

template<typename WeightRadius>
__global__ void filterSensitivity_kernel(int nebitword, gBitSAT<unsigned int> esat, int ereso, const float* g_sens, float* g_dst, float Rfilter, WeightRadius fr, const int* eidmap) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid >= nebitword) return;

	const unsigned int* ebit = esat._bitarray;
	const int* sat = esat._chunksat;

	unsigned int eword = ebit[tid];

	if (eword == 0) return;

	float R2 = Rfilter * Rfilter;
	int eidoffset = sat[tid];
	int ewordoffset = 0;
	for (int j = 0; j < BitCount<unsigned int>::value; j++) {
		if (read_gbit(eword, j)) {
			int bid = tid * BitCount<unsigned int>::value + j;
			int bpos[3] = { bid % ereso, bid % (ereso*ereso) / ereso, bid / (ereso * ereso) };
			int eid = eidoffset + ewordoffset;
			// traverse its spatial neighbors
			int R = Rfilter + 0.5;
			int L = -R;

			int npos[3];

			float weightSum = 0;
			double g_sum = 0;

			// DEBUG

			for (int x = L; x <= R; x++) {

				int x2 = x * x;
				npos[0] = bpos[0] + x;
				if (npos[0] < 0 || npos[0] >= ereso) continue;

				for (int y = L; y <= R; y++) {

					int y2 = y * y;
					npos[1] = bpos[1] + y;
					if (npos[1] < 0 || npos[1] >= ereso) continue;

					for (int z = L; z <= R; z++) {

						int z2 = z * z;
						npos[2] = bpos[2] + z;

						// spatial neighbor position
						if (npos[2] < 0 || npos[2] >= ereso) continue;

						float r2 = x2 + y2 + z2;
						if (r2 > R2) continue;

						// spatial neighbor bit id
						int n_bid = npos[0] + npos[1] * ereso + npos[2] * ereso * ereso;

						// spatial neighbor element id
						int n_eid = esat(n_bid);

						// if neighbor element is not valid
						if (n_eid == -1) continue;

						if (eidmap != nullptr) { n_eid = eidmap[n_eid]; }

						// weighted sum
						float w = fr(sqrtf(r2 / R2));

						g_sum += w * g_sens[n_eid];
						weightSum += w;

					}
				}
			} // traverse all spatial neighbor elements

			g_sum /= weightSum;

			if (eidmap != nullptr) eid = eidmap[eid];

			g_dst[eid] = g_sum;

			ewordoffset++;
		}
	}

	
}

void Grid::filterSensitivity(double radii)
{
	if (_layer != 0) return;
	
	size_t grid_size, block_size;

	make_kernel_param(&grid_size, &block_size, _gbuf.nword_ebits, 512);

	auto fr = [=] __device__(float r) {
		float r2 = r * r;
		return 1 - 6 * r2 + 8 * r2 * r - 3 * r2 *r2;
	};

	gBitSAT<unsigned int> esat(_gbuf.eActiveBits, _gbuf.eActiveChunkSum);

	float* g_sens_copy = (float*)getTempBuf(sizeof(float)* n_gselements);

	hipMemcpy(g_sens_copy, _gbuf.g_sens, sizeof(float) * n_gselements, hipMemcpyDeviceToDevice);

	init_array(_gbuf.g_sens, float{ 0 }, n_gselements);

	filterSensitivity_kernel << <grid_size, block_size >> > (_gbuf.nword_ebits, esat, _ereso, g_sens_copy, _gbuf.g_sens, radii, fr, _gbuf.eidmap);

	hipDeviceSynchronize();

	cuda_error_check;
}

__global__ void applyK_OTFA_kernel(int nv, devArray_t<double*, 3> u, devArray_t<double*, 3> f, float* rholist, bool use_support = true) {
	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	//int mode = gmode[0];

	__shared__ double KE[24][24];

	// load template matrix from constant memory to shared memory
	loadTemplateMatrix(KE);

	int vid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid >= nv) return;

	double KeU[3] = { 0. };

	double* pU[3] = { u[0],u[1],u[2] };

	float power = power_penalty[0];

	bool vifix = false;

	int viflag;

	if (!isValidNode(vid)) goto __writef;

	if (use_support) {
		viflag = gVflag[0][vid];
		vifix = viflag & grid::Grid::Bitmask::mask_supportnodes;
		if (vifix) {
			goto __writef;
		}
	}

	for (int e = 0; e < 8; e++) {

		int vi = 7 - e;

		int eid = gV2E[e][vid];

		if (eid == -1) continue;

		double penalty = powf(rholist[eid], power);

		for (int vj = 0; vj < 8; vj++) {
			int vjpos[3] = {
				vj % 2 + e % 2,
				vj % 4 / 2 + e % 4 / 2,
				vj / 4 + e / 4
			};
			int vj_lid = vjpos[0] + vjpos[1] * 3 + vjpos[2] * 9;
			int vj_vid = gV2V[vj_lid][vid];
			if (vj_vid == -1) continue;

			double u_vj[3] = { pU[0][vj_vid],pU[1][vj_vid],pU[2][vj_vid] };

			if (use_support) {
				int vjflag = gVflag[0][vj_vid];
				if (vjflag & grid::Grid::Bitmask::mask_supportnodes) {
					u_vj[0] = 0; u_vj[1] = 0; u_vj[2] = 0;
				}
			}

			for (int k = 0; k < 3; k++) {
				for (int j = 0; j < 3; j++) {
					KeU[k] += penalty * KE[k + vi * 3][j + vj * 3] * u_vj[j];
				}
			}
		}
	}

__writef:
	for (int i = 0; i < 3; i++) {
		if (use_support && vifix) {
			KeU[i] = gU[i][vid];
		}
		f[i][vid] = KeU[i];
	}
}

void Grid::applyK(double* u[3], double* f[3])
{
	use_grid();
	if (_layer == 0) {
		devArray_t<double*, 3> ulist{ u[0],u[1],u[2] };
		devArray_t<double*, 3> flist{ f[0],f[1],f[2] };
		size_t grid_size, block_size;
		make_kernel_param(&grid_size, &block_size, n_gsvertices, 512);
		applyK_OTFA_kernel << <grid_size, block_size >> > (n_gsvertices, ulist, flist, _gbuf.rho_e);
		hipDeviceSynchronize();
		cuda_error_check;
	}
	
}

void grid::Grid::resetDirchlet(double* v_dev[3])
{
	use_grid();
	if (_layer == 0) {
		devArray_t<double*, 3> vlist{ v_dev[0],v_dev[1],v_dev[2] };
		auto kernel = [=] __device__(int tid) {
			int flag = gVflag[0][tid];
			if ((flag & Grid::Bitmask::mask_supportnodes) && !(flag & Grid::Bitmask::mask_invalid)) {
				for (int i = 0; i < 3; i++) {
					vlist[i][tid] = 0;
				}
			}
		};	
		size_t grid_size, block_size;
		make_kernel_param(&grid_size, &block_size, n_gsvertices, 512);
		traverse_noret << <grid_size, block_size >> > (n_gsvertices, kernel);
		hipDeviceSynchronize();
		cuda_error_check;
	}
}

__global__ void cubeGridSetSolidVertices_kernel(int ereso, const unsigned int* ebits, unsigned int* vbits) {
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	int ne = ereso * ereso * ereso;
	int vreso = ereso + 1;
	int nv = vreso * vreso * vreso;

	if (tid >= nv) return;

	int vpos[3] = { tid % vreso, tid % (vreso * vreso) / vreso, tid / (vreso * vreso) };

	bool has_valid = false;
	for (int i = 0; i < 8; i++) {
		int epos[3] = { vpos[0] + i % 2 - 1, vpos[1] + (i % 4 / 2) - 1, vpos[2] + i / 4 - 1 };
		if (
			epos[0] >= ereso || epos[1] >= ereso || epos[2] >= ereso ||
			epos[0] < 0 || epos[1] < 0 || epos[2] < 0
			) continue;
		int eid = epos[0] + epos[1] * ereso + epos[2] * ereso * ereso;
		if (read_gbit(ebits, eid)) {
			has_valid = true;
			break;
		}
	}

	if (has_valid) {
		//set_gbit(vbits, tid);
		atomic_set_gbit(vbits, tid);
	}
}

void grid::cubeGridSetSolidVertices_g(int reso, const std::vector<unsigned int>& solid_ebit, std::vector<unsigned int>& solid_vbit)
{
	int vreso = reso + 1;
	int nv = pow(vreso, 3);
	int n_vword = snippet::Round< BitCount<unsigned int>::value >(nv) / BitCount<unsigned int>::value;

	unsigned int* g_ebits, *g_vbits;
	hipMalloc(&g_ebits, sizeof(unsigned int)*solid_ebit.size());
	hipMalloc(&g_vbits, n_vword * sizeof(unsigned int));

	hipMemcpy(g_ebits, solid_ebit.data(), sizeof(unsigned int) * solid_ebit.size(), hipMemcpyHostToDevice);
	init_array(g_vbits, (unsigned int)(0), n_vword);

	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, nv, 512);

	cubeGridSetSolidVertices_kernel << <grid_size, block_size >> > (reso, g_ebits, g_vbits);
	hipDeviceSynchronize();
	cuda_error_check;

	solid_vbit.resize(n_vword, 0);
	hipMemcpy(solid_vbit.data(), g_vbits, sizeof(unsigned int) * n_vword, hipMemcpyDeviceToHost);

	hipFree(g_ebits);
	hipFree(g_vbits);
}

__global__ void setSolidElementFromFineGrid_kernel(int finereso, const unsigned int* ebitsfine, unsigned int* ebitscoarse) {
	int tid = blockDim.x*blockIdx.x + threadIdx.x;

	int nvfine = finereso * finereso * finereso;

	int coarsereso = finereso >> 1;

	if (tid >= nvfine) return;

	// solid fine elements encountered
	if (read_gbit(ebitsfine, tid)) {
		// fine coarse element position
		int epos[3] = { tid %finereso, tid % (finereso*finereso) / finereso, tid / (finereso*finereso) };
		// coarse element position
		for (int i = 0; i < 3; i++) epos[i] >>= 1;
		// coarse element id
		int vcoarse = epos[0] + epos[1] * coarsereso + epos[2] * coarsereso * coarsereso;
		// set solid bit flag
		atomic_set_gbit(ebitscoarse, vcoarse);
	}
}

void grid::setSolidElementFromFineGrid_g(int finereso, const std::vector<unsigned int>& ebits_fine, std::vector<unsigned int>& ebits_coarse)
{
	int nefine = pow(finereso, 3);
	int necoarse = pow(finereso / 2, 3);
	int nword_coarse = snippet::Round<BitCount<unsigned int>::value>(necoarse) / BitCount<unsigned int>::value;

	unsigned int* g_fine, *g_coarse;
	hipMalloc(&g_fine, snippet::Round<BitCount<unsigned int>::value>(nefine) / 8);
	hipMalloc(&g_coarse, snippet::Round<BitCount<unsigned int>::value>(necoarse) / 8);

	hipMemcpy(g_fine, ebits_fine.data(), snippet::Round<BitCount<unsigned int>::value>(nefine) / 8, hipMemcpyHostToDevice);
	init_array(g_coarse, (unsigned int)(0), nword_coarse);

	int ne_fine = pow(finereso, 3);
	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, ne_fine, 512);
	setSolidElementFromFineGrid_kernel << <grid_size, block_size >> > (finereso, g_fine, g_coarse);
	hipDeviceSynchronize();
	cuda_error_check;

	ebits_coarse.resize(nword_coarse);

	hipMemcpy(ebits_coarse.data(), g_coarse, sizeof(unsigned int) * nword_coarse, hipMemcpyDeviceToHost);
	
	hipFree(g_fine);
	hipFree(g_coarse);
}

__global__ void wordReverse_kernel(size_t nword, unsigned int* g_words) {
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (tid >= nword) return;

	unsigned int word = g_words[tid];

	// reverse the word
	g_words[tid] = __brev(word);
}

void grid::wordReverse_g(size_t nword, unsigned int* wordlist)
{
	unsigned int* g_words;
	hipMalloc(&g_words, nword * sizeof(unsigned int));
	hipMemcpy(g_words, wordlist, nword * sizeof(unsigned int), hipMemcpyHostToDevice);
	
	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, nword, 512);
	wordReverse_kernel << <grid_size, block_size >> > (nword, g_words);
	hipDeviceSynchronize();
	cuda_error_check;

	hipMemcpy(wordlist, g_words, sizeof(unsigned int)*nword, hipMemcpyDeviceToHost);

	hipFree(g_words);
}

__global__ void setV2VCoarse_kernel(
	int nvword,
	int skip, int vresofine, gBitSAT<unsigned int> vsatfine,
	gBitSAT<unsigned int> vsatcoarse, devArray_t<int*, 8> v2vcoarse
) {
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (tid >= nvword) return;

	int vresocoarse = ((vresofine - 1) >> skip) + 1;
	int vresofine2 = vresofine * vresofine;
	int vresocoarse2 = vresocoarse * vresocoarse;
	int nvbfine = vresofine2 * vresofine;

	unsigned int coarseRatio = (1 << skip) ;
	double cr3 = coarseRatio * coarseRatio * coarseRatio;
	auto word = vsatfine._bitarray[tid];
	if (word == 0) return;
	for (int ji = 0; ji < grid::BitCount<unsigned int>::value; ji++) {
		if (!read_gbit(word, ji)) continue;
		int vbidfine = tid * grid::BitCount<unsigned int>::value + ji;
		if (vbidfine >= nvbfine) continue;
		int vposfine[3] = { vbidfine % vresofine, vbidfine / vresofine % vresofine, vbidfine / vresofine2 };
		int vposInE[3] = { (vposfine[0] % coarseRatio), (vposfine[1] % coarseRatio), (vposfine[2] % coarseRatio) };
		int vidfine = vsatfine[vbidfine];
		// traverse coarse element vertex
		for (int i = 0; i < 8; i++) {
			int vcoarsepos[3] = { i % 2 * coarseRatio, i % 4 / 2 * coarseRatio, i / 4 * coarseRatio };
			int wpos[3] = { abs(vcoarsepos[0] - vposInE[0]),abs(vcoarsepos[1] - vposInE[1]),abs(vcoarsepos[2] - vposInE[2]) };
			int vidcoarse = -1;
			if (wpos[0] < coarseRatio && wpos[1] < coarseRatio && wpos[2] < coarseRatio) {
				int vcoarsebitpos[3] = {
					(vposfine[0] - vposInE[0]) / coarseRatio + i % 2 ,
					(vposfine[1] - vposInE[1]) / coarseRatio + i % 4 / 2,
					(vposfine[2] - vposInE[2]) / coarseRatio + i / 4
				};
				int vcoarsebitid = vcoarsebitpos[0] + vcoarsebitpos[1] * vresocoarse + vcoarsebitpos[2] * vresocoarse2;
				vidcoarse = vsatcoarse(vcoarsebitid);
			}
			v2vcoarse[i][vidfine] = vidcoarse;
			//double weight = (coarseRatio - wpos[0]) * (coarseRatio - wpos[1]) * (coarseRatio - wpos[2]) / cr3;
		}
	}


}


void Grid::setV2VCoarse_g(
	int skip, int vresofine,
	grid::BitSAT<unsigned int>& vsatfine, grid::BitSAT<unsigned int>& vsatcoarse,
	int* v2vcoarse[8]
) {
	int nvword = vsatfine._bitArray.size();

	unsigned int* g_vbfine, *g_vbcoarse;
	int *g_vbfinesat, *g_vbcoarsesat;
	int* gv2vcoarse[8];

	for (int i = 0; i < 8; i++) {
		hipMalloc(&gv2vcoarse[i], sizeof(int) * vsatfine.total());
		init_array(gv2vcoarse[i], -1, vsatfine.total());
	}

	// copy vertex SAT from host to device
	gBitSAT<unsigned int> g_vsatfine(vsatfine._bitArray, vsatfine._chunkSat);
	gBitSAT<unsigned int> g_vsatcoarse(vsatcoarse._bitArray, vsatcoarse._chunkSat);

	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, nvword, 512);

	devArray_t<int*, 8> gv2vc_dev;
	for (int i = 0; i < 8; i++) gv2vc_dev[i] = gv2vcoarse[i];
	
	setV2VCoarse_kernel << <grid_size, block_size >> > (nvword, skip, vresofine, g_vsatfine, g_vsatcoarse, gv2vc_dev);
	hipDeviceSynchronize();
	cuda_error_check;

	for (int i = 0; i < 8; i++) {
		hipMemcpy(v2vcoarse[i], gv2vcoarse[i], sizeof(int) * vsatfine.total(), hipMemcpyDeviceToHost);
	}

	g_vsatfine.destroy();
	g_vsatcoarse.destroy();

	for (int i = 0; i < 8; i++) hipFree(gv2vcoarse[i]);

	cuda_error_check;
}

__global__ void setV2VFine_kernel(int nvcoarseword,
	int skip, int vresocoarse, gBitSAT<unsigned int> vsatfine,
	gBitSAT<unsigned int> vsatcoarse, devArray_t<int*, 27> v2vfine

) {
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if (tid >= nvcoarseword) return;

	int ncoarse = 1 << skip;

	int vresocoarse2 = vresocoarse * vresocoarse;

	int vresofine = (vresocoarse - 1) * ncoarse + 1;

	int nvbit = vresocoarse * vresocoarse * vresocoarse;

	unsigned int coarseword = vsatcoarse._bitarray[tid];

	if (coarseword == 0) return;

	for (int ji = 0; ji < BitCount<unsigned int>::value; ji++) {
		if (!read_gbit(coarseword, ji)) continue;
		int vcoarsebid = tid * BitCount<unsigned int>::value + ji;

		if (vcoarsebid >= nvbit) continue;

		int vidcoarse = vsatcoarse[vcoarsebid];

		int vcoarsepos[3] = { vcoarsebid % vresocoarse, vcoarsebid / vresocoarse % vresocoarse, vcoarsebid / vresocoarse2 };

		if (vcoarsepos[0] < 0 || vcoarsepos[0] >= vresocoarse ||
			vcoarsepos[1] < 0 || vcoarsepos[1] >= vresocoarse ||
			vcoarsepos[2] < 0 || vcoarsepos[2] >= vresocoarse
			) {
			continue;
		}

		int vfinepos[3] = { vcoarsepos[0] * ncoarse, vcoarsepos[1] * ncoarse, vcoarsepos[2] * ncoarse };

		for (int k = 0; k < 27; k++) {
			int vfineneipos[3] = { vfinepos[0] + k % 3 - 1, vfinepos[1] + (k / 3 % 3) - 1, vfinepos[2] + k / 9 - 1 };

			if (vfineneipos[0] < 0 || vfineneipos[0] >= vresofine ||
				vfineneipos[1] < 0 || vfineneipos[1] >= vresofine ||
				vfineneipos[2] < 0 || vfineneipos[2] >= vresofine
				) {
				continue;
			}

			int vfinenei_id = vfineneipos[0] + vfineneipos[1] * vresofine + vfineneipos[2] * vresofine * vresofine;

			//if (!read_gbit(vsatcoarse._bitarray, vfinenei_id)) continue;
			//int vidfine = vsatcoarse[vfinenei_id];
			int vidfine = vsatfine(vfinenei_id);

			v2vfine[k][vidcoarse] = vidfine;
		}
	}
}

void Grid::setV2VFine_g(
	int skip, int vresocoarse,
	grid::BitSAT<unsigned int>& vsatfine,
	grid::BitSAT<unsigned int>& vsatcoarse,
	int* v2vfine[27]
) {
	if (skip != 1) {
		printf("\033[31mV2VFine do not support non-dyadic coarse\033[0m\n");
		exit(-1);
	}

	int nvfineword = vsatfine._bitArray.size();
	int nvcoarseword = vsatcoarse._bitArray.size();

	unsigned int* g_vbfine, *g_vbcoarse;
	int *g_vbfinesat, *g_vbcoarsesat;

	devArray_t<int*, 27> gv2vfinelist;

	// copy host SAT to device
	gBitSAT<unsigned int> g_vsatfine(vsatfine._bitArray, vsatfine._chunkSat);
	gBitSAT<unsigned int> g_vsatcoarse(vsatcoarse._bitArray, vsatcoarse._chunkSat);

	for (int i = 0; i < 27; i++) {
		hipMalloc(&gv2vfinelist[i], sizeof(int) * vsatcoarse.total());
		init_array(gv2vfinelist[i], -1, vsatcoarse.total());
	}

	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, nvcoarseword, 512);

	setV2VFine_kernel << <grid_size, block_size >> > (nvcoarseword, skip, vresocoarse, g_vsatfine, g_vsatcoarse, gv2vfinelist);
	hipDeviceSynchronize();
	cuda_error_check;

	for (int i = 0; i < 27; i++) {
		hipMemcpy(v2vfine[i], gv2vfinelist[i], sizeof(int) * vsatcoarse.total(), hipMemcpyDeviceToHost);
	}

	g_vsatfine.destroy();
	g_vsatcoarse.destroy();
	gv2vfinelist.destroy();

	cuda_error_check;
}

__global__ void setV2VFineC_kernel(int nvcoarseword,int vresocoarse, gBitSAT<unsigned int> vsatfine2, gBitSAT<unsigned int> vsatcoarse, devArray_t<int*, 64> g_v2vfinec) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= nvcoarseword) return;
	
	int vresofinefine = (vresocoarse - 1) * 4 + 1;
	int vresofinefine2 = vresofinefine * vresofinefine;

	unsigned int word = vsatcoarse._bitarray[tid];

	int nvbit = vresocoarse * vresocoarse * vresocoarse;

	if (word == 0) return;

	for (int ji = 0; ji < BitCount<unsigned int>::value; ji++) {
		if (!read_gbit(word, ji)) continue;
		int vcoarsebid = tid * BitCount<unsigned int>::value + ji;
		if (vcoarsebid >= nvbit) continue;
		int vidcoarse = vsatcoarse[vcoarsebid];
		
		int vfinepos[3] = { vcoarsebid % vresocoarse * 4, vcoarsebid / vresocoarse % vresocoarse * 4, vcoarsebid / (vresocoarse * vresocoarse) * 4 };
		if (vfinepos[0] >= vresofinefine || vfinepos[1] >= vresofinefine || vfinepos[2] >= vresofinefine) continue;

		for (int k = 0; k < 64; k++) {
			int vfcpos[3] = { k % 4 * 2 + vfinepos[0] - 3, k / 4 % 4 * 2 + vfinepos[1] - 3, k / 16 * 2 + vfinepos[2] - 3 };

			if (vfcpos[0] < 0 || vfcpos[0] >= vresofinefine ||
				vfcpos[1] < 0 || vfcpos[1] >= vresofinefine ||
				vfcpos[2] < 0 || vfcpos[2] >= vresofinefine) {
				continue;
			}

			int vfcid = vfcpos[0] + vfcpos[1] * vresofinefine + vfcpos[2] * vresofinefine2;
			int vidfc = vsatfine2(vfcid);
			g_v2vfinec[k][vidcoarse] = vidfc;
		}
	}
}

void Grid::setV2VFineC_g(int vresocoarse, grid::BitSAT<unsigned int>& vsatfine2, grid::BitSAT<unsigned int>& vsatcoarse, int* v2vfinec[64])
{
	// copy host SAT to device
	gBitSAT<unsigned int> satfine2(vsatfine2._bitArray, vsatfine2._chunkSat);
	gBitSAT<unsigned int> satcoarse(vsatcoarse._bitArray, vsatcoarse._chunkSat);

	int nvcoarseword = vsatcoarse._bitArray.size();

	// allocate v2vfine buffer on device
	devArray_t<int*, 64> gv2vfc;
	for (int i = 0; i < 64; i++) {
		hipMalloc(&gv2vfc[i], sizeof(int) * vsatcoarse.total());
		init_array(gv2vfc[i], -1, vsatcoarse.total());
	}

	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, nvcoarseword, 512);

	// lauch kernel
	setV2VFineC_kernel << <grid_size, block_size >> > (nvcoarseword, vresocoarse, satfine2, satcoarse, gv2vfc);
	hipDeviceSynchronize();
	cuda_error_check;

	// copy result from device to host
	for (int i = 0; i < 64; i++) {
		hipMemcpy(v2vfinec[i], gv2vfc[i], sizeof(int)* vsatcoarse.total(), hipMemcpyDeviceToHost);
	}

	// free GPU memory
	satfine2.destroy();
	satcoarse.destroy();
	gv2vfc.destroy();
	cuda_error_check;
}

__global__ void setV2E_kernel(int nvword, int nvvalid, int nevalid, int vreso, gBitSAT<unsigned int> vrtsat, gBitSAT<unsigned int> elsat, devArray_t<int*, 8> v2elist) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ereso = vreso - 1;
	if (tid >= nvword) return;

	int nvbit = vreso * vreso * vreso;

	unsigned int vbitword = vrtsat._bitarray[tid];
	if (vbitword == 0) return;

	for (int ji = 0; ji < BitCount<unsigned int>::value; ji++) {
		if (!read_gbit(vbitword, ji)) continue;
		int vbitid = tid * BitCount<unsigned int>::value + ji;
		if (vbitid >= nvbit) continue;
		int vid = vrtsat[vbitid];
		int vpos[3] = { vbitid % vreso, vbitid / vreso % vreso, vbitid / (vreso*vreso) };
		for (int k = 0; k < 8; k++) {
			int epos[3] = { vpos[0] + k % 2 - 1,vpos[1] + k / 2 % 2 - 1,vpos[2] + k / 4 - 1 };

			if (epos[0] < 0 || epos[0] >= ereso ||
				epos[1] < 0 || epos[1] >= ereso ||
				epos[2] < 0 || epos[2] >= ereso) {
				continue;
			}

			int ebitid = epos[0] + epos[1] * ereso + epos[2] * ereso * ereso;

			int eid = elsat(ebitid);

			v2elist[k][vid] = eid;
		}
	}
}

void Grid::setV2E_g(int vreso, BitSAT<unsigned int>& vrtsat, BitSAT<unsigned int>& elsat, int* v2e[8])
{
	gBitSAT<unsigned int> g_vsat(vrtsat._bitArray, vrtsat._chunkSat);
	gBitSAT<unsigned int> g_esat(elsat._bitArray, elsat._chunkSat);
	devArray_t<int*, 8> g_v2e;
	for (int i = 0; i < 8; i++) {
		hipMalloc(&g_v2e[i], vrtsat.total() * sizeof(int));
		init_array(g_v2e[i], -1, vrtsat.total());
	}
	cuda_error_check;

	int n_vword = vrtsat._bitArray.size();

	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, n_vword, 512);
	setV2E_kernel << <grid_size, block_size >> > (n_vword, vrtsat.total(), elsat.total(), vreso, g_vsat, g_esat, g_v2e);
	hipDeviceSynchronize();
	cuda_error_check;

	for (int i = 0; i < 8; i++) {
		hipMemcpy(v2e[i], g_v2e[i], sizeof(int) * vrtsat.total(), hipMemcpyDeviceToHost);
	}
	
	g_v2e.destroy();
	g_vsat.destroy();
	g_esat.destroy();

	cuda_error_check;
}

__global__ void setV2V_kernel(int n_vword, int vreso, gBitSAT<unsigned int> vrtsat, devArray_t<int*, 27> g_v2v) {
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (tid >= n_vword) return;

	int nvbit = vreso * vreso * vreso;

	unsigned int vbitword = vrtsat._bitarray[tid];
	if (vbitword == 0) return;

	for (int ji = 0; ji < BitCount<unsigned int>::value; ji++) {
		if (!read_gbit(vbitword, ji)) continue;
		int vibid = tid * BitCount<unsigned int>::value + ji;
		if (vibid >= nvbit) continue;
		int viid = vrtsat[vibid];
		int vipos[3] = { vibid % vreso, vibid / vreso % vreso, vibid / (vreso * vreso) };

		for (int k = 0; k < 27; k++) {
			int vjpos[3] = { vipos[0] + k % 3 - 1,vipos[1] + k / 3 % 3 - 1,vipos[2] + k / 9 - 1 };

			if (vjpos[0] < 0 || vjpos[0] >= vreso ||
				vjpos[1] < 0 || vjpos[1] >= vreso ||
				vjpos[2] < 0 || vjpos[2] >= vreso) {
				continue;
			}

			int vjbid = vjpos[0] + vjpos[1] * vreso + vjpos[2] * vreso * vreso;

			int vjid = vrtsat(vjbid);

			g_v2v[k][viid] = vjid;
		}
	}
}

void Grid::setV2V_g(int vreso, BitSAT<unsigned int>& vrtsat, int* v2v[27])
{
	int n_vword = vrtsat._bitArray.size();

	devArray_t<int*, 27> g_v2v;
	for (int i = 0; i < 27; i++) {
		hipMalloc(&g_v2v[i], sizeof(int) * vrtsat.total());
		init_array(g_v2v[i], -1, vrtsat.total());
	}

	gBitSAT<unsigned int> g_vrtsat(vrtsat._bitArray, vrtsat._chunkSat);

	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, n_vword, 512);
	setV2V_kernel << <grid_size, block_size >> > (n_vword, vreso, g_vrtsat, g_v2v);
	hipDeviceSynchronize();
	cuda_error_check;

	for (int i = 0; i < 27; i++) {
		hipMemcpy(v2v[i], g_v2v[i], sizeof(int) * vrtsat.total(), hipMemcpyDeviceToHost);
	}

	g_v2v.destroy();
	g_vrtsat.destroy();
	cuda_error_check;
}

void Grid::init_rho(double rh0)
{
	init_array(_gbuf.rho_e, float(rh0), n_rho());
}

__global__ void computeNodePos_kernel(int n_word, int vreso, gBitSAT<unsigned int> vrtsat, devArray_t<double, 3> orig, double eh, devArray_t<double*, 3> pos) {
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (tid >= n_word) return;
	
	auto word = vrtsat._bitarray[tid];

	for (int ji = 0; ji < BitCount<unsigned int>::value; ji++) {
		if (!read_gbit(word, ji)) continue;
		int vbid = tid * BitCount<unsigned int>::value + ji;
		int vpos[3] = { vbid % vreso, vbid / vreso % vreso, vbid / vreso / vreso };
		int vid = vrtsat[vbid];
		for (int k = 0; k < 3; k++) {
			pos[k][vid] = orig[k] + eh * vpos[k];
		}
	}
}


void HierarchyGrid::getNodePos(Grid& g, std::vector<double>& p3host)
{
	int lay = g._layer;
	auto& vsat = vrtsatlist[lay];
	if (vsat.total() != g.n_vertices) printf("-- error on get node pos\n");
	int vreso = g._ereso + 1;
	devArray_t<double*, 3> p;
	for (int i = 0; i < 3; i++) {
		hipMalloc(&p[i], sizeof(double)*g.n_vertices);
	}

	double eh = elementLength() * (1 << g._layer);
	devArray_t<double, 3> orig;
	for (int i = 0; i < 3; i++) orig[i] = _gridlayer[0]->_box[0][i];

	int nword = vsat._bitArray.size();
	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, nword, 512);

	gBitSAT<unsigned int> vrtsat(vsat._bitArray, vsat._chunkSat);

	computeNodePos_kernel << <grid_size, block_size >> > (nword, vreso, vrtsat, orig, eh, p);
	hipDeviceSynchronize();
	cuda_error_check;


	p3host.resize(g.n_gsvertices * 3);
	double* gspos;
	hipMalloc(&gspos, sizeof(double) * g.n_gsvertices);
	for (int i = 0; i < 3; i++) {
		std::vector<double> hostpos;
		init_array(gspos, std::numeric_limits<double>::quiet_NaN(), g.n_gsvertices);
		int* vidmap = g._gbuf.vidmap;
		auto reorder = [=] __device__(int tid) {
			gspos[vidmap[tid]] = p[i][tid];
		};
		make_kernel_param(&grid_size, &block_size, g.n_vertices, 512);
		traverse_noret << <grid_size, block_size >> > (g.n_vertices, reorder);
		hipDeviceSynchronize();
		cuda_error_check;
		hostpos.resize(g.n_gsvertices);
		hipMemcpy(hostpos.data(), gspos, sizeof(double) * g.n_gsvertices, hipMemcpyDeviceToHost);
		for (int j = 0; j < g.n_gsvertices; j++) {
			p3host[j * 3 + i] = hostpos[j];
		}
	}


}

void HierarchyGrid::fillShell(void)
{
	_gridlayer[0]->use_grid();
	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, _gridlayer[0]->n_gsvertices, 512);
	int nv = _gridlayer[0]->n_gsvertices;
	float* rholist = _gridlayer[0]->_gbuf.rho_e;
	auto fillkernel = [=] __device__(int tid) {
		if (tid >= nv) return;
		int flag = gVflag[0][tid];
		if (flag & Grid::Bitmask::mask_invalid) return;
		int eid = gV2E[0][tid];
		if (eid == -1) return;
		int eflag = gEflag[0][eid];
		if (eflag & Grid::Bitmask::mask_shellelement) rholist[eid] = 1;
	};
	traverse_noret << <grid_size, block_size >> > (_gridlayer[0]->n_gsvertices, fillkernel);
	hipDeviceSynchronize();
	cuda_error_check;
}

float* Grid::getlexiEbuf(float* gs_src)
{
	float* dst = (float*)getTempBuf(sizeof(float) * n_elements);
	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, n_elements, 512);
	int* eidmap = _gbuf.eidmap;

	auto kernel = [=] __device__(int tid) {
		dst[tid] = gs_src[eidmap[tid]];
	};
	traverse_noret << <grid_size, block_size >> > (n_elements, kernel);
	hipDeviceSynchronize();
	cuda_error_check;
	return dst;
}

double* Grid::getlexiVbuf(double* gs_src)
{
	double* dst = (double*)getTempBuf(sizeof(double) * n_vertices);
	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, n_vertices, 512);
	int* vidmap = _gbuf.vidmap;

	auto kernel = [=] __device__(int tid) {
		dst[tid] = gs_src[vidmap[tid]];
	};
	traverse_noret << <grid_size, block_size >> > (n_vertices, kernel);
	hipDeviceSynchronize();
	cuda_error_check;
	return dst;
}

__global__ void apply_adjointK_kernel(int nv, float* rholist,
	devArray_t<double*, 3> usrc, devArray_t<double*, 3> fdst,
	gBitSAT<unsigned int> vloadsat, bool use_support, bool constrain_force
) {
	__shared__ double KE[24][24];

	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	loadTemplateMatrix(KE);

	if (tid >= nv) return;

	int vid = tid;

	int mode = gmode[0];

	// load flag and neighbor ids
	bool vfix[27], vload[27];
	int v2v[27];
	loadNeighborNodesAndFlags(vid, v2v, vfix, vload);

	double vitan[2][3] = { 0. };

	double KU[3] = { 0.,0.,0. };
	float power = power_penalty[0];
	for (int i = 0; i < 8; i++) {
		int eid = gV2E[i][vid];
		if (eid == -1) continue;
		double penalty = powf(rholist[eid], power);

		// vertex id in i-th neighbor element
		int vi = 7 - i;

		//double KeU[3] = { 0. };
		// traverse other vertex of neighbor element, and compute KeU
		for (int vj = 0; vj < 8; vj++) {
			int vjpos[3] = {
				vj % 2 + i % 2,
				vj % 4 / 2 + i % 4 / 2,
				vj / 4 + i / 4
			};
			int vj_lid = vjpos[0] + vjpos[1] * 3 + vjpos[2] * 9;
			int vj_vid = v2v[vj_lid];
			if (vj_vid == -1) {
				// DEBUG
				printf("-- error in update residual otfa\n");
				continue;
			}

			// check if vj is a load node, and load the tangent vectors if it its
			bool vjisload = vload[vj_lid];
			bool vjisfix = vfix[vj_lid];
			double vtan[2][3];
			if (vjisload) {
				int vjloadid = vloadsat(vj_vid); if (vjloadid == -1) printf("-- error on node %d\n", vj_vid);
				for (int k1 = 0; k1 < 2; k1++)
					for (int k2 = 0; k2 < 3; k2++) vtan[k1][k2] = gLoadtangent[k1][k2][vjloadid];
				// set viload if vj is vi
				if (vj_lid == 13) {
					for (int k1 = 0; k1 < 2; k1++)
						for (int k2 = 0; k2 < 3; k2++) vitan[k1][k2] = vtan[k1][k2];
				}
			}

			// fetch displacement
			double u[3] = { usrc[0][vj_vid],usrc[1][vj_vid],usrc[2][vj_vid] };

			if (vjisfix && use_support) {
				u[0] = 0; u[1] = 0; u[2] = 0;
			}

			// multiply N^T on u if vj is load node
			if (vjisload ) {
				if (constrain_force) {
					double Nu[3];
					for (int k = 0; k < 3; k++) Nu[k] = vtan[0][k] * u[0] + vtan[1][k] * u[1];
					for (int k = 0; k < 3; k++) u[k] = Nu[k];
				}
				else {
					u[0] = 0; u[1] = 0; u[2] = 0;
				}
			}

			for (int row = 0; row < 3; row++) {
				for (int col = 0; col < 3; col++) {
					KU[row] += penalty * KE[row + vi * 3][col + vj * 3] * u[col];
				}
			}
		}

	}
	// check whether vi is load node, multiply N if true
	if (vload[13] ) {
		if (constrain_force) {
			double ku[2] = { 0. };
			for (int k = 0; k < 3; k++) {
				ku[0] += vitan[0][k] * KU[k];
				ku[1] += vitan[1][k] * KU[k];
			}
			KU[0] = ku[0]; KU[1] = ku[1]; KU[2] = 0;
		}
		else {
			KU[0] = 0; KU[1] = 0; KU[2] = 0;
		}
	}

	// vi is fix
	if (vfix[13] && use_support) { KU[0] = 0; KU[1] = 0; KU[2] = 0; }

	for (int i = 0; i < 3; i++) { fdst[i][vid] = KU[i]; }
}

void grid::Grid::applyAjointK(double* usrc[3], double* fdst[3])
{
	use_grid();
	bool use_support = hasSupport();
	bool constrain_force = !isForceFree();
	devArray_t<double*, 3> us{ usrc[0],usrc[1],usrc[2] };
	devArray_t<double*, 3> fd{ fdst[0],fdst[1],fdst[2] };
	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, n_gsvertices, 512);
	apply_adjointK_kernel<<<grid_size,block_size>>>(n_gsvertices, _gbuf.rho_e,
		us, fd, vid2loadid, use_support, constrain_force
	);
	hipDeviceSynchronize();
	cuda_error_check;
}

void grid::Grid::v3_create(double* dstv[3])
{
	for (int i = 0; i < 3; i++) {
		hipMalloc(&dstv[i], sizeof(double) * n_gsvertices);
	}
}

bool grid::Grid::checkV2V(void)
{
	use_grid();
	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, n_gsvertices, 512);
	auto kernel = [=] __device__(int tid) {
		//int v2v[27];
		//loadNeighborNodes(tid, v2v);
		for (int i = 0; i < 8; i++) {
			int eid = gV2E[i][tid];
			if (eid == -1) continue;
			for (int j = 0; j < 8; j++) {
				int vjpos[3] = {
					i % 2 + j % 2,
					i / 2 % 2 + j / 2 % 2,
					i / 4 + j / 4
				};
				int vjlid = vjpos[0] + vjpos[1] * 3 + vjpos[2] * 9;
				if (gV2V[vjlid][tid] == -1) {
					printf("-- v[%d] e[%d](%d), v[%d]\n", tid, i, eid, vjlid);
				}
			}
		}
	};
	traverse_noret << <grid_size, block_size >> > (n_gsvertices, kernel);
	hipDeviceSynchronize();
	cuda_error_check;
	return false;
}

void grid::Grid::v3_pertub(double* v[3], double ratio)
{
	double oldnorm = v3_norm(v);

	// generate a pertubation
	devArray_t<double*, 3> pertub;
	pertub.create(n_gsvertices);
	v3_rand(pertub._data, -1, 1);
	double pertubnorm = v3_norm(pertub._data);
	v3_scale(pertub._data, 1.0 / pertubnorm * (oldnorm * ratio));

	// apply pertubation
	v3_add(v, 1, pertub._data);

	// new norm
	double newnorm = v3_norm(v);

	// scale new v3 to old norm 
	v3_scale(v, oldnorm / newnorm);

	// destroy temp buf
	pertub.destroy();
}

void grid::Grid::pertubForce(double ratio)
{
	// compute current force norm
	getForceSupport(_gbuf.F, getSupportForce());
	double** fsptr = getSupportForce();
	devArray_t<double*, 3> fs{ fsptr[0],fsptr[1],fsptr[2] };
	double oldnorm = norm(fs[0], fs[1], fs[2], n_loadnodes());

	// compute required noise norm
	double noisyNormRequare = oldnorm * ratio;

	// compute noise
	devArray_t<double*, 3> fsNoise;
	fsNoise.create(n_loadnodes());
	for (int i = 0; i < 3; i++) {
		randArray<double>(fsNoise._data, 3, n_loadnodes(), -1, 1);
	}
	double noiseNorm = norm(fsNoise[0], fsNoise[1], fsNoise[2], n_loadnodes());

	// add scaled noise
	double scaleRatio = noisyNormRequare / noiseNorm;
	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, n_loadnodes(), 512);
	for (int i = 0; i < 3; i++) {
		map<<<grid_size,block_size>>>(n_loadnodes(), [=]__device__(int tid) {
			for (int j = 0; j < 3; j++) {
				fs[j][tid] += fsNoise[j][tid] * scaleRatio;
			}
		});
	}
	hipDeviceSynchronize();
	cuda_error_check;

	setForceSupport(fsptr, _gbuf.F);
}


__global__ void elementCompliance_kernel(int nv, devArray_t<double*, 3> ulist, devArray_t<double*, 3> flist, float* rholist, float* clist) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	__shared__ double KE[24][24];

	loadTemplateMatrix(KE);

	if (tid >= nv) return;

	int v2v[27];

	loadNeighborNodes(tid, v2v);

	float power = power_penalty[0];

	for (int e = 0; e < 8; e++) {
		int vi = 7 - e;
		int eid = gV2E[e][tid];
		if (eid == -1) continue;
		float penal = powf(rholist[eid], power);
		double KeU[3] = { 0,0,0 };
		for (int vj = 0; vj < 8; vj++) {
			int vjpos[3] = { e % 2 + vj % 2, e / 2 % 2 + vj / 2 % 2, e / 4 + vj / 4 };
			int vjlid = vjpos[0] + vjpos[1] * 3 + vjpos[2] * 9;
			int vjid = v2v[vjlid];
			if (vjid == -1) continue;
			double Uj[3] = { gU[0][vjid], gU[1][vjid], gU[2][vjid] };
			for (int krow = 0; krow < 3; krow++) {
				for (int kcol = 0; kcol < 3; kcol++) {
					KeU[krow] += penal * KE[vi * 3 + krow][vj * 3 + kcol] * Uj[kcol];
				}
			}
		}

		double Ui[3] = { gU[0][tid],gU[1][tid],gU[2][tid] };

		double uKeu = Ui[0] * KeU[0] + Ui[1] * KeU[1] + Ui[2] * KeU[2];

		atomicAdd(clist + eid, uKeu);
	}
	
}

void grid::Grid::elementCompliance(double* u[3], double* f[3], float* dst)
{
	devArray_t<double*, 3> ulist, flist;
	for (int i = 0; i < 3; i++) {
		ulist[i] = u[i]; flist[i] = f[i];
	}

	init_array(dst, float{ 0 }, n_gselements);

	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, n_gsvertices, 512);
	elementCompliance_kernel << <grid_size, block_size >> > (n_gsvertices, ulist, flist, _gbuf.rho_e, dst);
	hipDeviceSynchronize();
	cuda_error_check;
}

double  grid::Grid::compliance(double* u[3], double* f[3])
{
	// create vector
	devArray_t<double*, 3> ku;
	ku.create(n_gsvertices);

	// compute f * K * u
	applyK(u, ku._data);

	double c = v3_dot(ku._data, f);
	ku.destroy();
	return c;
}

float grid::Grid::volumeRatio(void)
{
	cuda_error_check;
	float* tmp = (float*)getTempBuf(sizeof(float)* n_gselements / 100);
	float v = parallel_sum(_gbuf.rho_e, n_gselements);
	hipDeviceSynchronize();
	cuda_error_check;
	return v / n_gselements;
}

void grid::HierarchyGrid::test_kernels(void)
{
	_gridlayer[0]->use_grid();
	int nv = _gridlayer[0]->n_gsvertices;
	size_t grid_size, block_size;
	auto t0 = tictoc::getTag();
	float* rholist = _gridlayer[0]->_gbuf.rho_e;
	if (_mode == no_support_constrain_force_direction || _mode == no_support_free_force) {
		make_kernel_param(&grid_size, &block_size, nv, 512);
		update_residual_OTFA_NS_kernel << <grid_size, block_size >> > (nv, rholist);
	}
	else if (_mode == with_support_constrain_force_direction || _mode == with_support_free_force) {
		make_kernel_param(&grid_size, &block_size, nv, 256);
		update_residual_OTFA_WS_kernel << <grid_size, block_size >> > (nv, rholist);
	}
	hipDeviceSynchronize();
	cuda_error_check;
	auto t1 = tictoc::getTag();
	double t_duration = tictoc::Duration<tictoc::ms>(t0, t1);
	printf("[Routine1] time %6.2lf ms\n", t_duration);

	t0 = tictoc::getTag();
	if (_mode == no_support_constrain_force_direction || _mode == no_support_free_force) {
		make_kernel_param(&grid_size, &block_size, nv, 512);
		update_residual_OTFA_NS_kernel << <grid_size, block_size >> > (nv, rholist);
	}
	else if (_mode == with_support_constrain_force_direction || _mode == with_support_free_force) {
		make_kernel_param(&grid_size, &block_size, nv * 8, 32 * 8);
		update_residual_OTFA_WS_kernel_1 << <grid_size, block_size >> > (nv, rholist);
	}
	hipDeviceSynchronize();
	cuda_error_check;
	t1 = tictoc::getTag();
	t_duration = tictoc::Duration<tictoc::ms>(t0, t1);
	printf("[Routine2] time %6.2lf ms\n", t_duration);
}

double grid::Grid::densityDiscretiness(void)
{
	float* rholist = _gbuf.rho_e;
	float* pout = _gbuf.g_sens;
	auto disc = [=] __device__(int eid) {
		float rho = rholist[eid];
		pout[eid] = rho * (1 - rho);
	};
	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, n_gselements, 512);
	map<<<grid_size,block_size>>>(n_gselements, disc);
	hipDeviceSynchronize();
	cuda_error_check;

	double Md = parallel_sum_d(pout, n_gselements) / n_gselements;
	return Md;
}
